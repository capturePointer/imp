#include "hip/hip_runtime.h"
#include <imp/cuimgproc/cu_tvl1_denoising.cuh>

#include <iostream>

#include <hip/hip_runtime.h>

#include <imp/cuda_toolkit/hip/hip_vector_types.h>
#include <imp/core/pixel.hpp>
#include <imp/cucore/cu_texture.cuh>
#include <imp/cucore/cu_k_derivative.cuh>
#include <imp/cucore/cu_math.cuh>


namespace imp {
namespace cu {

//-----------------------------------------------------------------------------
__global__ void k_initTvL1Solver(Pixel32fC1* d_u, Pixel32fC1* d_u_prev, size_t stride_u,
                                 Pixel32fC2* d_p, size_t stride_p,
                                 imp::cu::Texture2D f_tex,
                                 size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float val = f_tex.fetch<float>(x,y);
    d_u[y*stride_u + x] = val;
    d_u_prev[y*stride_u + x] = val;
    d_p[y*stride_p + x] = Pixel32fC2(0.0f, 0.0f);
  }
}

//-----------------------------------------------------------------------------
__global__ void k_tvL1PrimalUpdate(
    Pixel32fC1* d_u, Pixel32fC1* d_u_prev, size_t stride_u,
    Texture2D f_tex, Texture2D u_tex, Texture2D p_tex,
    float lambda, float tau, float theta, size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float f = f_tex.fetch<float>(x,y);
    float u = u_tex.fetch<float>(x,y);
    float u_prev = u;
    float div = dpAd(p_tex, x, y, width, height);
    u += tau*div;

    float tau_lambda = tau*lambda;
    float residual = u - f;
    if (residual < -tau_lambda)
    {
      u += tau_lambda;
    }
    else if (residual > tau_lambda)
    {
      u -= tau_lambda;
    }
    else
    {
      u = f;
    }

    d_u[y*stride_u + x] = u;
    d_u_prev[y*stride_u + x] = u + theta*(u-u_prev);
  }
}

//-----------------------------------------------------------------------------
__global__ void k_tvL1DualUpdate(
    Pixel32fC2* d_p, size_t stride_p, Texture2D p_tex, Texture2D u_prev_tex,
    float sigma, size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float2 p = p_tex.fetch<float2>(x,y);
    float2 dp_u = dp(u_prev_tex, x, y);

    p = p + sigma*dp_u;
    p = p / max(1.0f, length(p));
    d_p[y*stride_p + x] = {p.x, p.y};
  }
}

//-----------------------------------------------------------------------------
//! @todo (MWE) move to a common place (also needed for other algorithms!)
__global__ void k_tvL1convertResult8uC1(Pixel8uC1* d_u, size_t stride_u,
                                    imp::cu::Texture2D u_tex,
                                    size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    d_u[y*stride_u + x] = static_cast<std::uint8_t>(
          255.0f * u_tex.fetch<float>(x,y));
  }
}

//#############################################################################

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void TvL1Denoising<Pixel, pixel_type>::init(const Size2u& size)
{
  Base::init(size);
  IMP_CUDA_CHECK();

  // setup textures
  f_tex_ = f_->genTexture(false, hipFilterModeLinear, hipAddressModeClamp,
                          (f_->bitDepth()==8) ? hipReadModeNormalizedFloat :
                                                hipReadModeElementType);
  u_tex_ = u_->genTexture(false, hipFilterModeLinear, hipAddressModeClamp,
                          hipReadModeElementType);
  u_prev_tex_ = u_prev_->genTexture(false, hipFilterModeLinear,
                                    hipAddressModeClamp, hipReadModeElementType);
  p_tex_ = p_->genTexture(false, hipFilterModeLinear, hipAddressModeClamp,
                          hipReadModeElementType);
  IMP_CUDA_CHECK();

  // init internal vars
  k_initTvL1Solver
      <<< dimGrid(), dimBlock() >>> (u_->data(), u_prev_->data(), u_->stride(),
                                     p_->data(), p_->stride(),
                                     *f_tex_, size_.width(), size_.height());
  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void TvL1Denoising<Pixel, pixel_type>::denoise(const ImageBasePtr& dst,
                                               const ImageBasePtr& src)
{
  if (params_.verbose)
  {
    std::cout << "[Solver @gpu] TvL1Denoising::denoise:" << std::endl;
  }

  if (src->size() != dst->size())
  {
    throw imp::cu::Exception("Input and output image are not of the same size.",
                             __FILE__, __FUNCTION__, __LINE__);
  }

  f_ = std::dynamic_pointer_cast<Image>(src);
  //! @todo (MWE) we could use dst for u_ if pixel_type is consistent

  if (size_ != f_->size())
  {
    this->init(f_->size());
  }

  // internal params
  float theta = 1.0f;
  //float L = sqrtf(8.0f);
  float sigma = 1.f/sqrtf(8.0f);
  float tau = 1.f/8.f;

  for(int iter = 0; iter < this->params_.max_iter; ++iter)
  {
    if (sigma < 1000.0f)
      theta = 1.f/sqrtf(1.0f+0.7f*this->params_.lambda*tau);
    else
      theta = 1.0f;

    if (params_.verbose)
    {
      std::cout << "(TvL1 solver) iter: " << iter << "; tau: " << tau
                << "; sigma: " << sigma << "; theta: " << theta << std::endl;
    }

    k_tvL1DualUpdate
        <<< dimGrid(), dimBlock() >>> (p_->data(), p_->stride(),
                                       *p_tex_, *u_prev_tex_,
                                       sigma, size_.width(), size_.height());

    k_tvL1PrimalUpdate
        <<< dimGrid(), dimBlock() >>> (u_->data(), u_prev_->data(), u_->stride(),
                                       *f_tex_, *u_tex_, *p_tex_,
                                       params_.lambda, tau, theta,
                                       size_.width(), size_.height());

    sigma /= theta;
    tau *= theta;
  }
  IMP_CUDA_CHECK();

  switch (dst->pixelType())
  {
  case PixelType::i8uC1:
  {
    std::shared_ptr<ImageGpu8uC1> u(std::dynamic_pointer_cast<ImageGpu8uC1>(dst));
    k_tvL1convertResult8uC1
        <<< dimGrid(), dimBlock() >>> (u->data(), u->stride(),
                                       *u_tex_, size_.width(), size_.height());
  }
  break;
  case PixelType::i32fC1:
  {
    std::shared_ptr<ImageGpu32fC1> u(std::dynamic_pointer_cast<ImageGpu32fC1>(dst));
    u_->copyTo(*u);
  }
  break;
  default:
    throw imp::cu::Exception("Unsupported PixelType.",
                             __FILE__, __FUNCTION__, __LINE__);
  }
  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void TvL1Denoising<Pixel, pixel_type>::print(std::ostream& os) const
{
  os << "TvL1 Denoising:" << std::endl;
  this->Base::print(os);
}

//=============================================================================
// Explicitely instantiate the desired classes
// (sync with typedefs at the end of the hpp file)
template class TvL1Denoising<imp::Pixel8uC1, imp::PixelType::i8uC1>;
template class TvL1Denoising<imp::Pixel32fC1, imp::PixelType::i32fC1>;

} // namespace cu
} // namespace imp
