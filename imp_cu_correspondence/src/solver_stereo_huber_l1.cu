#include "hip/hip_runtime.h"
#include <imp/cu_correspondence/solver_stereo_huber_l1.cuh>

#include <cmath>

#include <hip/hip_runtime.h>

#include <imp/cu_correspondence/variational_stereo_parameters.hpp>
#include <imp/cu_core/cu_image_gpu.cuh>
#include <imp/cu_imgproc/cu_image_filter.cuh>
#include <imp/cu_imgproc/cu_image_transform.cuh>
#include <imp/cu_core/cu_utils.hpp>
#include <imp/cu_core/cu_texture.cuh>
#include <imp/cu_core/cu_math.cuh>

#include "warped_gradients_kernel.cuh"
#include "solver_stereo_huber_l1_kernel.cuh"

namespace imp {
namespace cu {


//------------------------------------------------------------------------------
SolverStereoHuberL1::~SolverStereoHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
SolverStereoHuberL1::SolverStereoHuberL1(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level)
  : SolverStereoAbstract(params, size, level)
{
  u_.reset(new Image(size));
  u_prev_.reset(new Image(size));
  u0_.reset(new Image(size));
  pu_.reset(new Dual(size));
  ix_.reset(new Image(size));
  it_.reset(new Image(size));

  // and its textures
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
}

//------------------------------------------------------------------------------
void SolverStereoHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void SolverStereoHuberL1::init(const SolverStereoAbstract& rhs)
{
  const SolverStereoHuberL1* from =
      dynamic_cast<const SolverStereoHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(*from->u0_, *from->u_);
    imp::cu::resample(*u_, *from->u0_, imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(*u_, *from->u_, imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(*pu_, *from->pu_, imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void SolverStereoHuberL1::solve(std::vector<ImagePtr> images)
{
  if (params_->verbose > 0)
    std::cout << "StereoCtFWarpingLevelPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO

  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);
  u_->copyTo(*u_prev_);
  Fragmentation<16,16> frag(size_);

  // constants
  const float L = std::sqrt(8.f);
  const float tau = 1.f/L;
  const float sigma = 1.f/L;
  float lin_step = 0.5f;

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    u_->copyTo(*u0_);

    // compute warped spatial and temporal gradients
    k_warpedGradients
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (ix_->data(), it_->data(), ix_->stride(), ix_->width(), ix_->height(),
             *i1_tex_, *i2_tex_, *u0_tex_);

    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // dual kernel
      k_dualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(),
               size_.width(), size_.height(),
               params_->eps_u, sigma,
               *u_prev_tex_, *pu_tex_);

      // and primal kernel
      k_primalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               params_->lambda, tau, lin_step,
               *u_tex_, *u0_tex_, *pu_tex_, *ix_tex_, *it_tex_);
    } // iters
    lin_step /= 1.2f;

  } // warps
  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

