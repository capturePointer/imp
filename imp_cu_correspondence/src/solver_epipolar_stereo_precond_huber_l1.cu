#include "hip/hip_runtime.h"
#include <imp/cu_correspondence/solver_epipolar_stereo_precond_huber_l1.cuh>

#include <hip/hip_runtime.h>

#include <glog/logging.h>

#include <imp/cu_correspondence/variational_stereo_parameters.hpp>
#include <imp/cu_core/cu_image_gpu.cuh>
#include <imp/cu_imgproc/cu_image_filter.cuh>
#include <imp/cu_imgproc/cu_image_transform.cuh>
#include <imp/cu_core/cu_utils.hpp>
#include <imp/cu_core/cu_texture.cuh>
#include <imp/cu_core/cu_math.cuh>
#include <imp/cu_core/cu_k_setvalue.cuh>
#include <imp/cu_imgproc/edge_detectors.cuh>

#include "warped_gradients_kernel.cuh"
#include "solver_precond_huber_l1_kernel.cuh"
#include "solver_stereo_precond_huber_l1_weighted_kernel.cuh"
//#include "solver_epipolar_stereo_precond_huber_l1_kernel.cuh"

#define USE_EDGES 1

namespace imp {
namespace cu {

//------------------------------------------------------------------------------
SolverEpipolarStereoPrecondHuberL1::~SolverEpipolarStereoPrecondHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
SolverEpipolarStereoPrecondHuberL1::SolverEpipolarStereoPrecondHuberL1(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level,
    const std::vector<cu::PinholeCamera>& cams,
    const cu::Matrix3f& F,
    const cu::SE3<float>& T_mov_fix,
    const imp::cu::ImageGpu32fC1& depth_proposal,
    const imp::cu::ImageGpu32fC1& depth_proposal_sigma2)
  : SolverStereoAbstract(params, size, level)
{
  u_.reset(new ImageGpu32fC1(size));
  u_prev_.reset(new ImageGpu32fC1(size));
  u0_.reset(new ImageGpu32fC1(size));
  pu_.reset(new ImageGpu32fC2(size));
  q_.reset(new ImageGpu32fC1(size));
  iw_.reset(new ImageGpu32fC1(size));
  ix_.reset(new ImageGpu32fC1(size));
  it_.reset(new ImageGpu32fC1(size));
  xi_.reset(new ImageGpu32fC1(size));
  g_.reset(new ImageGpu32fC1(size));

  depth_proposal_.reset(new ImageGpu32fC1(size));
  depth_proposal_sigma2_.reset(new ImageGpu32fC1(size));

  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  q_tex_ =  q_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
  xi_tex_ =  xi_->genTexture(false, hipFilterModeLinear);
  g_tex_ =  g_->genTexture(false, hipFilterModeLinear);
  depth_proposal_tex_ =  depth_proposal_->genTexture(false, hipFilterModeLinear);
  depth_proposal_sigma2_tex_ =  depth_proposal_sigma2_->genTexture(false, hipFilterModeLinear);


  float scale_factor = std::pow(params->ctf.scale_factor, level);

  if (depth_proposal.size() == size)
  {
    LOG(INFO) << "Copy depth proposals " << depth_proposal.size() << " to level0 "
              << depth_proposal_->size();
    depth_proposal.copyTo(*depth_proposal_);
    depth_proposal_sigma2.copyTo(*depth_proposal_sigma2_);
  }
  else
  {
    float downscale_factor = 0.5f*((float)size.width()/(float)depth_proposal.width()+
                                   (float)size.height()/(float)depth_proposal.height());

    if (params_->verbose>2)
      LOG(INFO) << "depth proposal downscaled to level: " << level << "; size: " << size
                << "; downscale_factor: " << downscale_factor;

    imp::cu::resample(*depth_proposal_, depth_proposal);
    imp::cu::resample(*depth_proposal_sigma2_, depth_proposal_sigma2);
  }

  F_ = F;
  T_mov_fix_ = T_mov_fix;

  // assuming we receive the camera matrix for level0
  if  (level == 0)
  {
    cams_ = cams;
  }
  else
  {
    for (auto cam : cams)
    {
      cu::PinholeCamera scaled_cam = cam * scale_factor;
      cams_.push_back(scaled_cam);
    }
  }
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  q_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::init(const SolverStereoAbstract& rhs)
{
  const SolverEpipolarStereoPrecondHuberL1* from =
      dynamic_cast<const SolverEpipolarStereoPrecondHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(*from->u0_, *from->u_);
    imp::cu::resample(*u_, *from->u0_, imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(*u_, *from->u_, imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(*pu_, *from->pu_, imp::InterpolationMode::point, false);
  imp::cu::resample(*q_, *from->q_, imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::solve(std::vector<ImageGpu32fC1::Ptr> images)
{
  if (params_->verbose > 0)
    std::cout << "SolverEpipolarStereoPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO


  // image textures
  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);


  // constants
  constexpr float tau = 0.95f;
  constexpr float sigma = 0.95f;
  float lin_step = 0.5f;
  Fragmentation<16,16> frag(size_);
  constexpr float eta = 2.0f;

  // init
  u_->copyTo(*u_prev_);


  // check if a pointwise lambda is set in the parameters. otherwise we create
  // a local one to simplify kernel interfaces
  cu::ImageGpu32fC1::Ptr lambda;
  if (params_->lambda_pointwise)
    lambda = params_->lambda_pointwise;
  else
  {
    // make it as small as possible to reduce memory overhead. access is then
    // handled by the texture
    lambda.reset(new ImageGpu32fC1(1,1));
    lambda->setValue(params_->lambda);
  }
  lambda_tex_ = lambda->genTexture(false,hipFilterModePoint,
                                   hipAddressModeClamp, hipReadModeElementType);

  // compute edge weight
  imp::cu::naturalEdges(*g_, *images.at(0),
                        params_->edge_sigma, params_->edge_alpha, params_->edge_q);

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    u_->copyTo(*u0_);


    // compute warped spatial and temporal gradients
    k_warpedGradientsEpipolarConstraint
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (iw_->data(), ix_->data(), it_->data(), ix_->stride(), ix_->width(), ix_->height(),
             cams_.at(0), cams_.at(1), F_, T_mov_fix_,
             *i1_tex_, *i2_tex_, *u0_tex_,
             *depth_proposal_tex_);

    // compute preconditioner
#if USE_EDGES
    // compute preconditioner
    k_preconditionerWeighted
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             *lambda_tex_, *ix_tex_, *g_tex_);
#else
    k_preconditioner
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             params_->lambda, *ix_tex_);
#endif

    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
#if USE_EDGES
      // dual update kernel
      k_dualUpdateWeighted
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(), q_->data(), q_->stride(),
               size_.width(), size_.height(),
               params_->eps_u, sigma, eta, *lambda_tex_,
               *u_prev_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *it_tex_, *g_tex_);

      // and primal update kernel
      k_primalUpdateWeighted
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               tau, lin_step, *lambda_tex_,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_, *g_tex_);
#else
      // dual update kernel
      k_dualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(), q_->data(), q_->stride(),
               size_.width(), size_.height(),
               params_->eps_u, sigma, eta, *lambda_tex_,
               *u_prev_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *it_tex_);

      // and primal update kernel
      k_primalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               tau, lin_step, *lambda_tex_,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_);
#endif
    } // iters
    lin_step /= 1.2f;

  } // warps



  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

