#include <imp/cudepth/stereo_ctf_warping_level_precond_huber_l1.cuh>

#include <hip/hip_runtime.h>

#include <imp/cudepth/variational_stereo_parameters.hpp>
#include <imp/cucore/cu_image_gpu.cuh>
#include <imp/cuimgproc/cu_image_filter.cuh>
#include <imp/cuimgproc/cu_image_transform.cuh>
#include <imp/cucore/cu_utils.hpp>
#include <imp/cucore/cu_texture.cuh>
#include <imp/cucore/cu_math.cuh>

#include "cu_k_warped_gradients.cuh"
#include "cu_k_stereo_ctf_warping_level_precond_huber_l1.cuh"

namespace imp {
namespace cu {

//------------------------------------------------------------------------------
StereoCtFWarpingLevelPrecondHuberL1::~StereoCtFWarpingLevelPrecondHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
StereoCtFWarpingLevelPrecondHuberL1::StereoCtFWarpingLevelPrecondHuberL1(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level)
  : SolverStereoAbstract(params, size, level)
{
  u_.reset(new Image(size));
  u_prev_.reset(new Image(size));
  u0_.reset(new Image(size));
  pu_.reset(new Dual(size));
  q_.reset(new Image(size));
  ix_.reset(new Image(size));
  it_.reset(new Image(size));
  xi_.reset(new Image(size));

  // and its textures
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  q_tex_ =  q_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
  xi_tex_ =  xi_->genTexture(false, hipFilterModeLinear);
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelPrecondHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  q_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelPrecondHuberL1::init(const SolverStereoAbstract& rhs)
{
  const StereoCtFWarpingLevelPrecondHuberL1* from =
      dynamic_cast<const StereoCtFWarpingLevelPrecondHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(*from->u0_, *from->u_);
    imp::cu::resample(*u_, *from->u0_, imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(*u_, *from->u_, imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(*pu_, *from->pu_, imp::InterpolationMode::point, false);
  imp::cu::resample(*q_, *from->q_, imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelPrecondHuberL1::solve(std::vector<ImagePtr> images)
{
  if (params_->verbose > 0)
    std::cout << "StereoCtFWarpingLevelPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO

  // image textures
  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);
  u_->copyTo(*u_prev_);
  Fragmentation<16,16> frag(size_);

  // constants
  constexpr float tau = 0.95f;
  constexpr float sigma = 0.95f;
  float lin_step = 0.5f;

  // precond
  constexpr float eta = 2.0f;

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    u_->copyTo(*u0_);

    // compute warped spatial and temporal gradients
    k_warpedGradients
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (ix_->data(), it_->data(), ix_->stride(), ix_->width(), ix_->height(),
             *i1_tex_, *i2_tex_, *u0_tex_);

    // compute preconditioner
    k_preconditioner
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             params_->lambda, *ix_tex_);


    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // dual update kernel
      k_dualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(), q_->data(), q_->stride(),
               size_.width(), size_.height(),
               params_->lambda, params_->eps_u, sigma, eta,
               *u_prev_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *it_tex_);

      // and primal update kernel
      k_primalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               params_->lambda, tau, lin_step,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_);
    } // iters
    lin_step /= 1.2f;

  } // warps
  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

