#include "hip/hip_runtime.h"
/*
 * Copyright (c) ICG. All rights reserved.
 *
 * Institute for Computer Graphics and Vision
 * Graz University of Technology / Austria
 *
 *
 * This software is distributed WITHOUT ANY WARRANTY; without even
 * the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE.  See the above copyright notices for more information.
 *
 *
 * Project     : ImageUtilities
 * Module      : Core
 * Class       : none
 * Language    : C/CUDA
 * Description : CUDA kernels for core functions
 *
 * Author     : Manuel Werlberger
 * EMail      : werlberger@icg.tugraz.at
 *
 */

#ifndef IUCORE_CONVERT_CU
#define IUCORE_CONVERT_CU

#include <hip/hip_vector_types.h>
#include "coredefs.h"
#include "memorydefs.h"
#include "iutextures.cuh"

namespace iuprivate {


/* ***************************************************************************
 *  CUDA KERNELS
 * ***************************************************************************/

//-----------------------------------------------------------------------------
/** convert kernel 32f_C3 -> 32f_C4 (float3 -> float4)
 */
__global__ void cuConvertC3ToC4Kernel(const float3* src, size_t src_stride, int src_width, int src_height,
                                      float4* dst, size_t dst_stride, int dst_width, int dst_height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    float3 val=src[src_c];
    dst[dst_c] =  make_float4(val.x, val.y, val.z, 1.0f);
  }
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 32f_C3 (float4 -> float3)
 */
__global__ void cuConvertC4ToC3Kernel(const float4* src, size_t src_stride, int src_width, int src_height,
                                      float3* dst, size_t dst_stride, int dst_width, int dst_height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    float4 val=src[src_c];
    dst[dst_c] = make_float3(val.x, val.y, val.z);
  }
}


//-----------------------------------------------------------------------------
/** convert kernel 8u_C1 -> 32f_C1 (unsigned char -> float)
 */
__global__ void cuConvert8uC1To32fC1Kernel(const unsigned char *src, size_t src_stride, int src_width, int src_height,
                                           float* dst, size_t dst_stride, int dst_width, int dst_height, float mul_constant,
                                           float add_constant)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}


/** convert kernel 8u_C3 -> 32f_C4 (unsigned char -> float)
 */
__global__ void cuConvert8uC3To32fC4Kernel(const unsigned char *src, size_t src_pitch, int src_width, int src_height,
                                           float4* dst, size_t dst_stride, int dst_width, int dst_height, float mul_constant,
                                           float add_constant)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_pitch + x*3;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    dst[dst_c] = make_float4(src[src_c]/255.0f, src[src_c+1]/255.0f, src[src_c+2]/255.0f, 1.0f);// * mul_constant + add_constant;
  }
}


//-----------------------------------------------------------------------------
/** convert kernel 32f_C1 -> 8u_C1 (float -> unsigned char)
 */
__global__ void cuConvert32fC1To8uC1Kernel(const float* src, size_t src_stride, int src_width, int src_height,
                                           unsigned char* dst, size_t dst_stride, int dst_width, int dst_height, float mul_constant,
                                           unsigned char add_constant)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 8u_C4 (float4 -> unsigned char4)
 */
__global__ void cuConvert32fC4To8uC4Kernel(const float4* src, size_t src_stride, int src_width, int src_height,
                                           uchar4* dst, size_t dst_stride, int dst_width, int dst_height, float mul_constant,
                                           unsigned char add_constant)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    float4 val = src[src_c];
    uchar4 res;
    res.x = val.x * mul_constant + add_constant;
    res.y = val.y * mul_constant + add_constant;
    res.z = val.z * mul_constant + add_constant;
    res.w = val.w * mul_constant + add_constant;
    dst[dst_c] = res;
  }
}

//-----------------------------------------------------------------------------
/** convert kernel rgb -> hsv
 */
__global__ void cuConvertRGBToHSVKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height, bool normalize)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    float R = in.x;
    float G = in.y;
    float B = in.z;

    float Ma = IUMAX(R, IUMAX(G, B));
    float mi = IUMIN(R, IUMIN(G, B));
    float C = Ma-mi;

    // Hue
    float H = 0.0f;
    if (C != 0.0f)
    {
      if (Ma == R)
        H = fmod((G - B)/C, 6.0f);
      if (Ma == G)
        H = (B - R)/C + 2.0f;
      if (Ma == B)
        H = (R - G)/C + 4.0f;
    }

    H *= 60.0f;

    // Value
    float V = Ma;

    // Saturation
    float S = 0.0f;
    if (C != 0.0f)
      S = C/V;

    if (H < 0.0f)
      H += 360.0f;

    // Normalize
    if (normalize)
      H /= 360.0f;

    // Write Back
    dst[c] = make_float4(H, S, V, in.w);
  }
}

//-----------------------------------------------------------------------------
/** convert kernel hsv -> rgb
 */
__global__ void cuConvertHSVToRGBKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height, bool denormalize)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    float H = in.x;
    float S = in.y;
    float V = in.z;

    float4 rgb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    //    float C = V*S;

    //    // Denormalize
    //    if (denormalize)
    //      H = H*360.0f;

    //    // RGB
    //    H /= 60.0f;
    //    float X = C*(1.0f - abs(fmod(H, 2.0f) - 1.0f));


    //    if (H >= 0.0f)
    //    {
    //      if (H < 1.0f)
    //        rgb = make_float4(C, X, 0.0f, 0.0f);
    //      else if (H < 2.0f)
    //        rgb = make_float4(X, C, 0.0f, 0.0f);
    //      else if (H < 3.0f)
    //        rgb = make_float4(0.0f, C, X, 0.0f);
    //      else if (H < 4.0f)
    //        rgb = make_float4(0.0f, X, C, 0.0f);
    //      else if (H < 5.0f)
    //        rgb = make_float4(X, 0.0f, C, 0.0f);
    //      else if (H <= 6.0f)
    //        rgb = make_float4(C, 0.0f, X, 0.0f);
    //    }

    //    float m = V-C;
    //    rgb += m;

    if (S == 0)
    {
      rgb = make_float4(V, V, V, in.w);
      dst[c] = rgb;
      return;
    }

     H /= 60.0f;
     int i = floor(H);
     float f = H-i;
     float p = V*(1.0f - S);
     float q = V*(1.0f - S*f);
     float t = V*(1.0f - S*(1.0f-f));

     if (i == 0)
       rgb = make_float4(V, t, p, in.w);
     else if (i == 1)
       rgb = make_float4(q, V, p, in.w);
     else if (i == 2)
       rgb = make_float4(p, V, t, in.w);
     else if (i == 3)
       rgb = make_float4(p, q, V, in.w);
     else if (i == 4)
       rgb = make_float4(t, p, V, in.w);
     else if (i == 5)
       rgb = make_float4(V, p, q, in.w);



    // Write Back
    rgb.w = in.w;
    dst[c] = rgb;
  }
}


//-----------------------------------------------------------------------------
/** convert kernel rgb -> lab
 */
__global__ void cuConvertRGBToLABKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height, bool isNormalized)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    if (!isNormalized)
      in /= 255.0f;
    
    float R = in.x;
    float G = in.y;
    float B = in.z;

    
    // convert to XYZ
    float4 XYZ;
    XYZ.x = 0.4124564f*R + 0.3575761f*G + 0.1804375f*B;
    XYZ.y = 0.2126729f*R + 0.7151522f*G + 0.0721750f*B;
    XYZ.z = 0.0193339f*R + 0.1191920f*G + 0.9503041f*B;
    
    // normalize for D65 white point
    XYZ.x /= 0.950456f;
    XYZ.z /= 1.088754f;
    
    float cubeRootX, cubeRootY, cubeRootZ;
    const float T1 = 216/24389.0f;
    const float T2 = 24389/27.0f;
    
   if (XYZ.x > T1)
      cubeRootX = cbrtf(XYZ.x);
    else
      cubeRootX = (T2 * XYZ.x + 16) / 116;
    
    if (XYZ.y > T1)
      cubeRootY = cbrtf(XYZ.y);
    else
      cubeRootY = (T2 * XYZ.y + 16) / 116;
    
    if (XYZ.z > T1)
      cubeRootZ = cbrtf(XYZ.z);
    else
      cubeRootZ = (T2 * XYZ.z + 16) / 116;
    
    
    
    dst[c] = make_float4(116*cubeRootY-16, 500*(cubeRootX-cubeRootY), 200*(cubeRootY-cubeRootZ), in.w);
  }
}


//-----------------------------------------------------------------------------
/** convert kernel lab -> rgb
 */
__global__ void cuConvertLABToRGBKernel(const float4* src, float4* dst, size_t stride,
                                        int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    
    float L = in.x;
    float a = in.y;
    float b = in.z;

    
    // convert to XYZ
    const float T1 = cbrtf(216/24389.0f);
    const float fy = (L+16) / 116.0f;
    
    float4 XYZ;
    if (L > 8)
      XYZ.y = fy*fy*fy;
    else
      XYZ.y = L / (24389/27.0f);
    
    float fx = a/500.0f + fy;
    if (fx > T1)
      XYZ.x = fx*fx*fx;
    else
      XYZ.x = (116*fx-16) / (24389/27.0f);
    
    float fz = fy - b/200.0f;
    if (fz > T1)
      XYZ.z = fz*fz*fz;
    else
      XYZ.z = (116*fz-16) / (24389/27.0f);
    
    
    // Normalize for D65 white point
    XYZ.x *= 0.950456f;
    XYZ.z *= 1.088754f;
    
    float4 rgb;
    rgb.x = 3.2404542f*XYZ.x + -1.5371385f*XYZ.y + -0.4985314f*XYZ.z;
    rgb.y = -0.9692660f*XYZ.x + 1.8760108f*XYZ.y + 0.0415560f*XYZ.z;
    rgb.z = 0.0556434f*XYZ.x + -0.2040259f*XYZ.y + 1.0572252f*XYZ.z;
    rgb.w = in.w;
    
    dst[c] = rgb;
  }
}



/* ***************************************************************************
 *  CUDA WRAPPERS
 * ***************************************************************************/

//-----------------------------------------------------------------------------
/** convert kernel 32f_C3 -> 32f_C4 (float3 -> float4)
 */
void cuConvert(const iu::ImageGpu_32f_C3* src, const IuRect& src_roi,
                   iu::ImageGpu_32f_C4* dst, const IuRect& dst_roi)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvertC3ToC4Kernel<<<dimGrid, dimBlock>>>(src->data(src_roi.x, src_roi.y), src->stride(),
                                               src_roi.width, src_roi.height,
                                               dst->data(dst_roi.x, dst_roi.y), dst->stride(),
                                               dst_roi.width, dst_roi.height);

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 32f_C3 (float4 -> float3)
 */
void cuConvert(const iu::ImageGpu_32f_C4* src, const IuRect& src_roi,
                   iu::ImageGpu_32f_C3* dst, const IuRect& dst_roi)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvertC4ToC3Kernel<<<dimGrid, dimBlock>>>(src->data(src_roi.x, src_roi.y), src->stride(),
                                               src_roi.width, src_roi.height,
                                               dst->data(dst_roi.x, dst_roi.y), dst->stride(),
                                               dst_roi.width, dst_roi.height);

  IU_CUDA_CHECK();
}


//-----------------------------------------------------------------------------
void cuConvert_8u_32f(const iu::ImageGpu_8u_C1* src, const IuRect& src_roi,
                          iu::ImageGpu_32f_C1* dst, const IuRect& dst_roi, float mul_constant,
                          float add_constant)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvert8uC1To32fC1Kernel<<<dimGrid, dimBlock>>>(src->data(src_roi.x, src_roi.y),
                                                    src->stride(), src_roi.width, src_roi.height,
                                                    dst->data(dst_roi.x, dst_roi.y),
                                                    dst->stride(), dst_roi.width, dst_roi.height,
                                                    mul_constant, add_constant);

  IU_CUDA_CHECK();
}


//-----------------------------------------------------------------------------
void cuConvert_8u_32f_C3C4(const iu::ImageGpu_8u_C3* src, const IuRect& src_roi,
                          iu::ImageGpu_32f_C4* dst, const IuRect& dst_roi, float mul_constant,
                          float add_constant)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));
  
  cuConvert8uC3To32fC4Kernel<<<dimGrid, dimBlock>>>(&(src->data(src_roi.x, src_roi.y)->x),
                                                    src->pitch(), src_roi.width, src_roi.height,
                                                    dst->data(dst_roi.x, dst_roi.y),
                                                    dst->stride(), dst_roi.width, dst_roi.height,
                                                    mul_constant, add_constant);

  IU_CUDA_CHECK();
}


//-----------------------------------------------------------------------------
void cuConvert_32f_8u(const iu::ImageGpu_32f_C1* src, const IuRect& src_roi,
                          iu::ImageGpu_8u_C1* dst, const IuRect& dst_roi, float mul_constant,
                          unsigned char add_constant)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvert32fC1To8uC1Kernel<<<dimGrid, dimBlock>>>(src->data(src_roi.x, src_roi.y), src->stride(),
                                                    src_roi.width, src_roi.height,
                                                    dst->data(dst_roi.x, dst_roi.y),
                                                    dst->stride(), dst_roi.width, dst_roi.height,
                                                    mul_constant, add_constant);

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
void cuConvert_32f_8u(const iu::ImageGpu_32f_C4* src, const IuRect& src_roi,
                          iu::ImageGpu_8u_C4* dst, const IuRect& dst_roi, float mul_constant,
                          unsigned char add_constant)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvert32fC4To8uC4Kernel<<<dimGrid, dimBlock>>>(src->data(src_roi.x, src_roi.y),
                                                    src->stride(), src_roi.width, src_roi.height,
                                                    dst->data(dst_roi.x, dst_roi.y),
                                                    dst->stride(), dst_roi.width,
                                                    dst_roi.height, mul_constant, add_constant);

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
void cuConvert_rgb_to_hsv(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst,
                              bool normalize)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertRGBToHSVKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height(), normalize);

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
void cuConvert_hsv_to_rgb(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst,
                              bool denormalize)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertHSVToRGBKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height(), denormalize);

  IU_CUDA_CHECK();
}


//-----------------------------------------------------------------------------
void cuConvert_rgb_to_lab(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst, bool isNormalized)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertRGBToLABKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height(), isNormalized);

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
void cuConvert_lab_to_rgb(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertLABToRGBKernel<<<dimGrid, dimBlock>>>(src->data(), dst->data(), src->stride(),
                                                 src->width(), src->height());

  IU_CUDA_CHECK();
}



} // namespace iuprivate

#endif // IUCORE_CONVERT_CU
