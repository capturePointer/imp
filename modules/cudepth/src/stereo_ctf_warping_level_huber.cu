#include <imp/cudepth/stereo_ctf_warping_level_huber.cuh>

#include <hip/hip_runtime.h>

#include <imp/cudepth/variational_stereo_parameters.hpp>
#include <imp/cucore/cu_image_gpu.cuh>
#include <imp/cuimgproc/cu_image_filter.cuh>
#include <imp/cuimgproc/cu_image_transform.cuh>
#include <imp/cucore/cu_utils.hpp>
#include <imp/cucore/cu_texture.cuh>

namespace imp {
namespace cu {

//------------------------------------------------------------------------------
StereoCtFWarpingLevelHuber::~StereoCtFWarpingLevelHuber()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
StereoCtFWarpingLevelHuber::StereoCtFWarpingLevelHuber(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level)
  : StereoCtFWarpingLevel(params, size, level)
{
  u_.reset(new Image(size));
  u_prev_.reset(new Image(size));
  u0_.reset(new Image(size));
  pu_.reset(new Dual(size));
  q_.reset(new Image(size));
  ix_.reset(new Image(size));
  it_.reset(new Image(size));

  // and its textures
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  q_tex_ =  q_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);

}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelHuber::init()
{
  u_->setValue(0.0f);
  u_prev_->setValue(0.0f);
  u0_->setValue(0.0f);
  pu_->setValue(0.0f);
  q_->setValue(0.0f);
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelHuber::init(const StereoCtFWarpingLevel& rhs)
{
  const StereoCtFWarpingLevelHuber* from =
      dynamic_cast<const StereoCtFWarpingLevelHuber*>(&rhs);

  float inv_sf = params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(from->u0_.get(), from->u_.get());
    imp::cu::resample(u_.get(), from->u0_.get(), imp::InterpolationMode::linear, false);
  }
  imp::cu::resample(u_.get(), from->u_.get(), imp::InterpolationMode::linear, false);
  *u_ *= inv_sf;

  imp::cu::resample(pu_.get(), from->pu_.get(), imp::InterpolationMode::linear, false);
  imp::cu::resample(q_.get(), from->q_.get(), imp::InterpolationMode::linear, false);
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelHuber::solve(std::vector<ImagePtr> images)
{
  std::cout << "StereoCtFWarpingLevelHuber: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // image textures

  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);

  // constants
//  constexpr float tau = 0.95f;
//  constexpr float sigma = 0.95f;
  float lin_step = 0.5f;

  // precond
//  constexpr float eta = 2.0f;

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    u_->copyTo(*u0_);

    // warping + gradients computation
    // TODO

    // compute preconditioner
    // TODO

    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // solve dual kernel
      // TODO


      // and primal kernel
      // TODO

    } // iters
    lin_step /= 1.2f;

  } // warps
  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

