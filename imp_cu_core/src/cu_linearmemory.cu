#include <imp/cu_core/cu_linearmemory.cuh>

#include <imp/cu_core/cu_exception.hpp>
#include <imp/cu_core/cu_utils.hpp>
#include <imp/cu_core/cu_k_setvalue.cuh>


namespace imp {
namespace cu {

//-----------------------------------------------------------------------------
template<typename Pixel>
LinearMemory<Pixel>::LinearMemory(const size_t& length)
  : LinearMemoryBase(length)
  , data_(Memory::alloc(this->length()))
{
}

//-----------------------------------------------------------------------------
template<typename Pixel>
LinearMemory<Pixel>::LinearMemory(const imp::cu::LinearMemory<Pixel>& from)
  : imp::cu::LinearMemory<Pixel>(from.length())
{
  if (from.data() == 0)
  {
    throw imp::cu::Exception("'from' data not valid", __FILE__, __FUNCTION__, __LINE__);
  }
  this->copyFrom(from);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
LinearMemory<Pixel>::LinearMemory(const imp::LinearMemory<Pixel>& from)
  : imp::cu::LinearMemory<Pixel>(from.length())
{
  if (from.data() == 0)
  {
    throw imp::cu::Exception("'from' data not valid", __FILE__, __FUNCTION__, __LINE__);
  }
  this->copyFrom(from);
}


////-----------------------------------------------------------------------------
//template<typename Pixel>
//LinearMemory<Pixel>::LinearMemory(Pixel* host_data,
//                                  const size_t& length,
//                                  bool use_ext_data_pointer)
//  : LinearMemoryBase(length)
//{
//  if (host_data == nullptr)
//  {
//    throw imp::cu::Exception("input data not valid", __FILE__, __FUNCTION__, __LINE__);
//  }

//  if(use_ext_data_pointer)
//  {
//    // This uses the external data pointer and stores it as a 'reference':
//    // memory won't be managed by us!
//    auto dealloc_nop = [](Pixel*) { ; };
//    data_ = std::unique_ptr<Pixel, Deallocator>(
//          host_data, Deallocator(dealloc_nop));
//  }
//  else
//  {
//    // allocates an internal data pointer and copies the external data it.
//    data_.reset(CuMemory::alignedAlloc(this->length()));
//    std::copy(host_data, host_data+length, data_.get());
//  }
//}

//-----------------------------------------------------------------------------
template<typename Pixel>
Pixel* LinearMemory<Pixel>::data()
{
  return data_.get();
}

//-----------------------------------------------------------------------------
template<typename Pixel>
const Pixel* LinearMemory<Pixel>::data() const
{
  return reinterpret_cast<const Pixel*>(data_.get());
}

//-----------------------------------------------------------------------------
template<typename Pixel>
auto LinearMemory<Pixel>::cuData() -> decltype(imp::cu::toCudaVectorType(this->data()))
{
  return imp::cu::toCudaVectorType(this->data());
}

//-----------------------------------------------------------------------------
template<typename Pixel>
auto LinearMemory<Pixel>::cuData() const -> decltype(imp::cu::toConstCudaVectorType(this->data()))
{
  return imp::cu::toConstCudaVectorType(this->data());
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::setValue(const Pixel& value)
{
  if (sizeof(Pixel) == 1)
  {
    hipMemset((void*)this->data(), (int)value.c[0], this->bytes());
  }
  else
  {
    // fragmentation
    cu::Fragmentation<32,1> frag(this->length());

    // todo add roi to kernel!
    imp::cu::k_setValue
        <<< frag.dimGrid, frag.dimBlock
        >>> (this->data(), this->length(), value);
  }
  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyTo(imp::cu::LinearMemory<Pixel>& dst)
{
  if (dst.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->bytes() != dst.bytes())
    IMP_THROW_EXCEPTION("source and destination array are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(dst.data(), this->data(), this->bytes(), hipMemcpyDeviceToDevice);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyFrom(const imp::cu::LinearMemory<Pixel>& from)
{
  if (from.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->bytes() != from.bytes())
    IMP_THROW_EXCEPTION("source and destination array are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(this->data(), from.data(), from.bytes(), hipMemcpyDeviceToDevice);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyTo(imp::LinearMemory<Pixel>& dst)
{
  if (dst.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->bytes() != dst.bytes())
    IMP_THROW_EXCEPTION("source and destination array are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(dst.data(), this->data(), this->bytes(), hipMemcpyDeviceToHost);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}

//-----------------------------------------------------------------------------
template<typename Pixel>
void LinearMemory<Pixel>::copyFrom(const imp::LinearMemory<Pixel>& from)
{
  if (from.data() == 0 || !data_)
    IMP_THROW_EXCEPTION("'from' or 'to' data is not valid");
  if (this->bytes() != from.bytes())
    IMP_THROW_EXCEPTION("source and destination array are of different length (byte length checked)");

  const hipError_t cu_err =
      hipMemcpy(this->data(), from.data(), from.bytes(), hipMemcpyHostToDevice);

  if (cu_err != hipSuccess)
    IMP_CU_THROW_EXCEPTION("hipMemcpy returned error code", cu_err);
}


//=============================================================================
// Explicitely instantiate the desired classes
template class LinearMemory<imp::Pixel8uC1>;
template class LinearMemory<imp::Pixel8uC2>;
template class LinearMemory<imp::Pixel8uC3>;
template class LinearMemory<imp::Pixel8uC4>;

template class LinearMemory<imp::Pixel16uC1>;
template class LinearMemory<imp::Pixel16uC2>;
template class LinearMemory<imp::Pixel16uC3>;
template class LinearMemory<imp::Pixel16uC4>;

template class LinearMemory<imp::Pixel32sC1>;
template class LinearMemory<imp::Pixel32sC2>;
template class LinearMemory<imp::Pixel32sC3>;
template class LinearMemory<imp::Pixel32sC4>;

template class LinearMemory<imp::Pixel32fC1>;
template class LinearMemory<imp::Pixel32fC2>;
template class LinearMemory<imp::Pixel32fC3>;
template class LinearMemory<imp::Pixel32fC4>;

} // namespace cu
} // namespace imp
