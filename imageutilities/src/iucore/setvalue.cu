#include "hip/hip_runtime.h"
/*
 * Copyright (c) ICG. All rights reserved.
 *
 * Institute for Computer Graphics and Vision
 * Graz University of Technology / Austria
 *
 *
 * This software is distributed WITHOUT ANY WARRANTY; without even
 * the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE.  See the above copyright notices for more information.
 *
 *
 * Project     : ImageUtilities
 * Module      : Core
 * Class       : none
 * Language    : C
 * Description : Implementation of CUDA functions to set a value to GPU memory
 *
 * Author     : Manuel Werlberger
 * EMail      : werlberger@icg.tugraz.at
 *
 */


#ifndef IUCORE_SETVALUE_CU
#define IUCORE_SETVALUE_CU

#include "coredefs.h"
#include "memorydefs.h"
#include "iucutil.h"

namespace iuprivate {

/* ****************************************************************************
 *  1D
 * ****************************************************************************/

//-----------------------------------------------------------------------------
// kernel: 1D set values; 1D
template<class T>
__global__ void cuSetValueKernel(T value, T* dst, int length)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if(x<length)
  {
    dst[x] = value;
  }
}

//-----------------------------------------------------------------------------
// wrapper: set values; 1D; 8-bit
/** Sets values of 1D linear gpu memory.
 * \param value The pixel value to be set.
 * \param buffer Pointer to the buffer
 */
void cuSetValue(const unsigned char& value, iu::LinearDeviceMemory_8u_C1* dst)
{
  // fragmentation
  const unsigned int block_width = 512;
  dim3 dimBlock(block_width, 1, 1);
  dim3 dimGrid(iu::divUp(dst->length(), dimBlock.x), 1);

  cuSetValueKernel <<< dimGrid, dimBlock >>> (
      value, dst->data(), dst->length());

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
// wrapper: set values; 1D; 32-bit
/** Sets values of 1D linear gpu memory.
 * \param value The pixel value to be set.
 * \param buffer Pointer to the buffer
 */
void cuSetValue(const int& value, iu::LinearDeviceMemory_32s_C1* dst)
{
  // fragmentation
  const unsigned int block_width = 512;
  dim3 dimBlock(block_width, 1, 1);

  int numChunks = iu::divUp(iu::divUp(dst->length(), dimBlock.x), 65535);
  if (numChunks > 1)
  {
    for (int i=0; i < numChunks-1; i++)
    {
      unsigned int globalPos = i*65535*dimBlock.x;        // calculate start index of current chunk
      dim3 dimGrid(65535, 1);                             // max grid dimension
    
      cuSetValueKernel <<< dimGrid, dimBlock >>> (         // kernel writes 65535*dimBlock.x elements
	value, dst->data(globalPos), 65535*dimBlock.x);
    }
    // calculate start index of last chunk
    unsigned int lastChunkStart = (numChunks-1)*65535*dimBlock.x;
    
    // determine grid size
    dim3 dimGrid(iu::divUp(dst->length()-lastChunkStart, dimBlock.x), 1);
    cuSetValueKernel <<< dimGrid, dimBlock >>> (       // kernel writes remaining elements
	value, dst->data(lastChunkStart), dst->length()-lastChunkStart);
  }
  else       // memory is smaller than 65535*dimBlock.x elements
  {
    dim3 dimGrid(iu::divUp(dst->length(), dimBlock.x), 1);
    
    cuSetValueKernel <<< dimGrid, dimBlock >>> (
	value, dst->data(), dst->length());
  }
  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
// wrapper: set values; 1D; 32-bit
/** Sets values of 1D linear gpu memory.
 * \param value The pixel value to be set.
 * \param buffer Pointer to the buffer
 */
void cuSetValue(const float& value, iu::LinearDeviceMemory_32f_C1* dst)
{
  // fragmentation
  const unsigned int block_width = 512;
  dim3 dimBlock(block_width, 1, 1);
  
  // FIXXXME: apply this also to the other setValue() functions
  
  // if memory is too long to be set with a single kernel calculate number of required
  // chunks.
  // Max grid dimension according to cuda pragramming guide: 65535
  int numChunks = iu::divUp(iu::divUp(dst->length(), dimBlock.x), 65535);
  if (numChunks > 1)
  {
    for (int i=0; i < numChunks-1; i++)
    {
      unsigned int globalPos = i*65535*dimBlock.x;        // calculate start index of current chunk
      dim3 dimGrid(65535, 1);                             // max grid dimension
    
      cuSetValueKernel <<< dimGrid, dimBlock >>> (         // kernel writes 65535*dimBlock.x elements
	value, dst->data(globalPos), 65535*dimBlock.x);
    }
    // calculate start index of last chunk
    unsigned int lastChunkStart = (numChunks-1)*65535*dimBlock.x;
    
    // determine grid size
    dim3 dimGrid(iu::divUp(dst->length()-lastChunkStart, dimBlock.x), 1);
    cuSetValueKernel <<< dimGrid, dimBlock >>> (       // kernel writes remaining elements
	value, dst->data(lastChunkStart), dst->length()-lastChunkStart);
  }
  else       // memory is smaller than 65535*dimBlock.x elements
  {
    dim3 dimGrid(iu::divUp(dst->length(), dimBlock.x), 1);
    
    cuSetValueKernel <<< dimGrid, dimBlock >>> (
	value, dst->data(), dst->length());
  }

  IU_CUDA_CHECK();
}


/* ****************************************************************************
 *  2D
 * ****************************************************************************/

//-----------------------------------------------------------------------------
// kernel: 2D set values; multi-channel
template<class T>
__global__ void cuSetValueKernel(T value, T* dst, size_t stride,
                                 int xoff, int yoff, int width, int height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int c = y*stride+x;

  // add xoff for checks after calculating the output pixel location c
  x+=xoff;
  y+=yoff;

  if(x>=0 && y>=0 && x < xoff+width && y < yoff+height)
  {
    dst[c] = value;
  }
}

//-----------------------------------------------------------------------------
// templated wrapper: set value; 2D;
template<typename PixelType, class Allocator, IuPixelType _pixel_type>
void cuSetValueTemplate(const PixelType &value,
                            iu::ImageGpu<PixelType, Allocator, _pixel_type> *dst,
                            const IuRect& roi, bool useMemset = false)
{
  if (useMemset && roi.width == dst->width() && roi.height == dst->height() &&
      roi.x == 0 && roi.y == 0)
  {
    // if value = 0 use memset() which is a lot faster than the kernel call
    hipMemset2D(dst->data(), dst->pitch(), 0, dst->width()*sizeof(PixelType), dst->height());
    //printf("using fast memset\n");
  }
  else
  {
    // fragmentation
    const unsigned int block_size = 16;
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid(iu::divUp(roi.width, dimBlock.x),
		iu::divUp(roi.height, dimBlock.y));
  
    cuSetValueKernel <<< dimGrid, dimBlock >>> (
	value, dst->data(roi.x, roi.y), dst->stride(),
	roi.x, roi.y, roi.width, roi.height);
    //printf("using kernel memset\n");
  }
  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
// specialized wrapper: set values (single value); 2D; 8-bit;
void cuSetValue(const unsigned char& value, iu::ImageGpu_8u_C1 *dst, const IuRect &roi)
{ 
  if (value == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const uchar2& value, iu::ImageGpu_8u_C2 *dst, const IuRect &roi)
{ 
  if (value.x == 0 && value.y == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}
void cuSetValue(const uchar3& value, iu::ImageGpu_8u_C3 *dst, const IuRect &roi)
{ 
  if (value.x == 0 && value.y == 0 && value.z == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}
void cuSetValue(const uchar4& value, iu::ImageGpu_8u_C4 *dst, const IuRect &roi)
{ 
  if (value.x == 0 && value.y == 0 && value.z == 0 && value.w == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}
// wrapper: set values (single value); 2D; 32-bit;
void cuSetValue(const int& value, iu::ImageGpu_32s_C1 *dst, const IuRect &roi)
{ return cuSetValueTemplate(value, dst, roi); }
// wrapper: set values (single value); 2D; 32-bit;
void cuSetValue(const float& value, iu::ImageGpu_32f_C1 *dst, const IuRect &roi)
{ 
  if (value == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}
void cuSetValue(const float2& value, iu::ImageGpu_32f_C2 *dst, const IuRect &roi)
{ 
  if (value.x == 0 && value.y == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}
void cuSetValue(const float3& value, iu::ImageGpu_32f_C3 *dst, const IuRect &roi)
{
  if (value.x == 0 && value.y == 0 && value.z == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}
void cuSetValue(const float4& value, iu::ImageGpu_32f_C4 *dst, const IuRect &roi)
{ 
  if (value.x == 0 && value.y == 0 && value.z == 0 && value.w == 0)
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi); 
}


/* ****************************************************************************
 *  3D
 * ****************************************************************************/

//-----------------------------------------------------------------------------
// kernel: 3D set values; multi-channel
template<class T>
__global__ void cuSetValueKernel(T value, T* dst, size_t stride, size_t slice_stride,
                                 int xoff, int yoff, int zoff, int roi_width, int roi_height,
                                 int roi_depth, int width, int height, int depth)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  const int c =  y*stride+x;

  x+=xoff;
  y+=yoff;

  if(x>=0 && y>=0 && x<min(width,xoff+roi_width) && y<min(height,yoff+roi_height))
  {
    for(int z = 0; z<min(depth-zoff, roi_depth); ++z)
      dst[c+z*slice_stride] = value;
  }
}

//-----------------------------------------------------------------------------
// templated wrapper: set values (single value); 3D; ...
template<typename PixelType, class Allocator, IuPixelType _pixel_type>
void cuSetValueTemplate(const PixelType &value,
                            iu::VolumeGpu<PixelType, Allocator, _pixel_type> *dst,
                            const IuCube& roi, bool useMemset = false)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x),
               iu::divUp(roi.height, dimBlock.y));

  if (useMemset)
  {
    // dammit cuda, y u need width in bytes and height & depth in elements???
    hipPitchedPtr pp = make_hipPitchedPtr(dst->data(),
                                            dst->pitch(), dst->width()*sizeof(PixelType),
                                            dst->height());
    hipExtent ex = make_hipExtent(roi.width*sizeof(PixelType), roi.height, roi.depth);
    hipMemset3D(pp, 0, ex);
  }
  else
  {
    cuSetValueKernel <<< dimGrid, dimBlock >>> (
      value, dst->data(roi.x, roi.y, roi.z), dst->stride(), dst->slice_stride(),
      roi.x, roi.y, roi.z, roi.width, roi.height, roi.depth, dst->width(), dst->height(),
      dst->depth());
  }

  IU_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
// specialized wrapper: set values (single value); 3D; 8-bit;
void cuSetValue(const unsigned char& value, iu::VolumeGpu_8u_C1 *dst, const IuCube &roi)
{ return cuSetValueTemplate(value, dst, roi); }
void cuSetValue(const uchar2& value, iu::VolumeGpu_8u_C2 *dst, const IuCube &roi)
{ return cuSetValueTemplate(value, dst, roi); }
void cuSetValue(const uchar4& value, iu::VolumeGpu_8u_C4 *dst, const IuCube &roi)
{ return cuSetValueTemplate(value, dst, roi); }
// wrapper: set values (single value); 3D; 32-bit;
void cuSetValue(const float& value, iu::VolumeGpu_32f_C1 *dst, const IuCube &roi)
{
  if (value == 0 && roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const unsigned int& value, iu::VolumeGpu_32u_C1 *dst, const IuCube &roi)
{
  if (value == 0 && roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const uint2& value, iu::VolumeGpu_32u_C2 *dst, const IuCube &roi)
{
  if (value.x == 0 && value.y == 0 &&  roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const uint4& value, iu::VolumeGpu_32u_C4 *dst, const IuCube &roi)
{
  if (value.x == 0 && value.y == 0 && value.z == 0 && value.w == 0 && roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const int& value, iu::VolumeGpu_32s_C1 *dst, const IuCube &roi)
{
  if (value == 0 && roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const int2& value, iu::VolumeGpu_32s_C2 *dst, const IuCube &roi)
{
  if (value.x == 0 && value.y == 0 &&  roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const int4& value, iu::VolumeGpu_32s_C4 *dst, const IuCube &roi)
{
  if (value.x == 0 && value.y == 0 && value.z == 0 && value.w == 0 && roi.x == 0 && roi.y == 0 && roi.z == 0 && roi.width == dst->width() &&
      roi.height == dst->height() && roi.depth == dst->depth())
    return cuSetValueTemplate(value, dst, roi, true);
  else
    return cuSetValueTemplate(value, dst, roi);
}
void cuSetValue(const float2& value, iu::VolumeGpu_32f_C2 *dst, const IuCube &roi)
{ return cuSetValueTemplate(value, dst, roi); }
void cuSetValue(const float4& value, iu::VolumeGpu_32f_C4 *dst, const IuCube &roi)
{ return cuSetValueTemplate(value, dst, roi); }
void cuSetValue(const unsigned short& value, iu::VolumeGpu_16u_C1 *dst, const IuCube &roi)
{ return cuSetValueTemplate(value, dst, roi); }



} // namespace iuprivate

#endif // IUCORE_SETVALUE_CU
