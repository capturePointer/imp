#include "hip/hip_runtime.h"
/*
 * Copyright (c) ICG. All rights reserved.
 *
 * Institute for Computer Graphics and Vision
 * Graz University of Technology / Austria
 *
 *
 * This software is distributed WITHOUT ANY WARRANTY; without even
 * the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE.  See the above copyright notices for more information.
 *
 *
 * Project     : ImageUtilities
 * Module      : GUI
 * Class       : qgl
 * Language    : CUDA
 * Description : Implementation of device functions for the qgl.
 *
 * Author     : Manuel Werlberger
 * EMail      : werlberger@icg.tugraz.at
 *
 */


#ifndef IUPRIVATE_QGL_IMAGE_GPU_WIDGET_CU
#define IUPRIVATE_QGL_IMAGE_GPU_WIDGET_CU

#include "iucutil.h"
#include "iudefs.h"

namespace iuprivate {

texture<unsigned char,  2, hipReadModeElementType> tex_qgl_image_8u_C1;
texture<uchar4, 2, hipReadModeElementType> tex_qgl_image_8u_C4;
texture<float,  2, hipReadModeElementType> tex_qgl_image_32f_C1;
texture<float4, 2, hipReadModeElementType> tex_qgl_image_32f_C4;


/** Kernel to copy image data into OpenGL PBO. */
__global__ void cuCopyImageToPboKernel_8u_C1(uchar4* dst, int width, int height)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int c = y*width+x;
  //Add half of a texel to always address exact texel centers
  const float xx = (float)x + 0.5f;
  const float yy = (float)y + 0.5f;

  if(x<width && y<height)
  {
    unsigned char val = tex2D(tex_qgl_image_8u_C1, xx, yy);
    dst[c] = make_uchar4(val, val, val, 1.0f);
  }
}

/** Kernel to copy image data into OpenGL PBO. */
__global__ void cuCopyImageToPboKernel_8u_C4(uchar4* dst, int width, int height)
{
  unsigned long x = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned long y = blockDim.y * blockIdx.y + threadIdx.y;

  const int c = y*width+x;
  //Add half of a texel to always address exact texel centers
  const float xx = (float)x + 0.5f;
  const float yy = (float)y + 0.5f;

  if(x<width && y<height)
  {
    uchar4 val = tex2D(tex_qgl_image_8u_C4, xx, yy);
    dst[c] = val;
  }
}


/** Kernel to copy image data into OpenGL PBO. */
__global__ void cuCopyImageToPboKernel_32f_C1(uchar4* dst, int width, int height,
                                              float min=0.0f, float max=1.0f)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int c = y*width+x;
  //Add half of a texel to always address exact texel centers
  const float xx = (float)x + 0.5f;
  const float yy = (float)y + 0.5f;

  if(x<width && y<height)
  {
    float val = tex2D(tex_qgl_image_32f_C1, xx, yy);
    //-min/(max-min)
    //val = val * 255.0f;
    val = 255.0f / (max-min) * (val-min);
    dst[c] = make_uchar4(val, val, val, 1.0f);

  }
}

/** Kernel to copy image data into OpenGL PBO. */
__global__ void cuCopyImageToPboKernel_32f_C4(uchar4* dst, int width, int height)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int c = y*width+x;
  //Add half of a texel to always address exact texel centers
  const float xx = (float)x + 0.5f;
  const float yy = (float)y + 0.5f;

  if(x<width && y<height)
  {
    float4 val = tex2D(tex_qgl_image_32f_C4, xx, yy);
    dst[c] = make_uchar4(val.x*255.0f, val.y*255.0f, val.z*255.0f, val.w*255.0f);
  }
}

IuStatus cuCopyImageToPbo(iu::Image* image, unsigned int num_channels,
                          unsigned int bit_depth, uchar4 *dst,
                          float min=0.0f, float max=1.0f)
{
  // device fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(image->width(), dimBlock.x),
               iu::divUp(image->height(), dimBlock.y));

  if(bit_depth == 8)
  {
    if(num_channels == 1)
    {
      iu::ImageGpu_8u_C1* img = reinterpret_cast<iu::ImageGpu_8u_C1*>(image);
      hipChannelFormatDesc channel_desc = hipCreateChannelDesc<unsigned char>();
      hipBindTexture2D(0, &tex_qgl_image_8u_C1, img->data(), &channel_desc,
                        img->width(), img->height(), img->pitch());
      cuCopyImageToPboKernel_8u_C1 <<< dimGrid, dimBlock >>> (dst, img->width(), img->height());
      hipUnbindTexture(tex_qgl_image_8u_C1);
    }
    else
    {
      iu::ImageGpu_8u_C4* img = reinterpret_cast<iu::ImageGpu_8u_C4*>(image);
      hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();
      hipBindTexture2D(0, &tex_qgl_image_8u_C4, img->data(), &channel_desc,
                        img->width(), img->height(), img->pitch());
      cuCopyImageToPboKernel_8u_C4 <<< dimGrid, dimBlock >>> (dst, img->width(), img->height());
      hipUnbindTexture(tex_qgl_image_8u_C4);
    }
  }
  else
  {
    if(num_channels == 1)
    {
      iu::ImageGpu_32f_C1* img = reinterpret_cast<iu::ImageGpu_32f_C1*>(image);
      hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
      hipBindTexture2D(0, &tex_qgl_image_32f_C1, img->data(), &channel_desc,
                        img->width(), img->height(), img->pitch());
      cuCopyImageToPboKernel_32f_C1 <<< dimGrid, dimBlock >>> (dst, img->width(), img->height(), min, max);
      hipUnbindTexture(tex_qgl_image_32f_C1);
    }
    else
    {
      iu::ImageGpu_32f_C4* img = reinterpret_cast<iu::ImageGpu_32f_C4*>(image);
      hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
      hipBindTexture2D(0, &tex_qgl_image_32f_C4, img->data(), &channel_desc,
                        img->width(), img->height(), img->pitch());
      cuCopyImageToPboKernel_32f_C4 <<< dimGrid, dimBlock >>> (dst, img->width(), img->height());
      hipUnbindTexture(tex_qgl_image_32f_C4);
    }
  }

  return iu::checkCudaErrorState();
}




} // namespace iuprivate


#endif // IUPRIVATE_QGL_IMAGE_GPU_WIDGET_CU
