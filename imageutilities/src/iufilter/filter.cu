#include "hip/hip_runtime.h"
/*
 * Copyright (c) ICG. All rights reserved.
 *
 * Institute for Computer Graphics and Vision
 * Graz University of Technology / Austria
 *
 *
 * This software is distributed WITHOUT ANY WARRANTY; without even
 * the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE.  See the above copyright notices for more information.
 *
 *
 * Project     : ImageUtilities
 * Module      : Filter
 * Class       : none
 * Language    : CUDA
 * Description : Definition of CUDA wrappers for filter functions on Npp images
 *
 * Author     : Manuel Werlberger
 * EMail      : werlberger@icg.tugraz.at
 *
 */

#ifndef IUPRIVATE_FILTER_CU
#define IUPRIVATE_FILTER_CU

#include <float.h>
#include <iucutil.h>
#include <iucore/iutextures.cuh>
#include <iucore/copy.h>
#include <iucore/setvalue.h>

#include "filterbspline_kernels.cu"

#include "filter.cuh"

namespace iuprivate {

// ----------------------------------------------------------------------------
// kernel: median filter; 32-bit; 1-channel
__global__ void  cuFilterMedian3x3Kernel_32f_C1(float* dst, const size_t stride,
                                                const int xoff, const int yoff,
                                                const int width, const int height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int oc = y*stride+x;

  x += xoff;
  y += yoff;

  float xx = x+0.5f;
  float yy = y+0.5f;

  // shared stuff
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  // we have a 3x3 kernel, so our width of the shared memory (shp) is blockDim.x + 2!
  const int shp = blockDim.x + 2;
  const int shc = (threadIdx.y+1) * shp + (threadIdx.x+1);
  extern __shared__ float sh_in[];

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    // Load input 3x3 block into shared memory
    {
      // for each thread: copy the data of the current input position to shared mem
      sh_in[shc] = tex2D(tex1_32f_C1__, xx, yy);

      // Note: the FLT_MAX prevents us from overemphasizing the border pixels if they are outliers!

      /////////////////////////////////////////////////////////////////////////////
      // boundary conditions
      /////////////////////////////////////////////////////////////////////////////
      if (x == 0) // at left image border
      {
        if (y == 0)
          sh_in[shc-shp-1] = FLT_MAX; // left-upper corner (image)
        else if (ty == 1)
          sh_in[shc-shp-1] = tex2D(tex1_32f_C1__, xx, yy-1.0f); // left-upper corner (block)

        sh_in[shc-1] = sh_in[shc];     // left border (image)

        if (y == height-1)
          sh_in[shc+shp-1] = FLT_MAX; // left-lower corner (image)
        else if (ty == blockDim.y)
          sh_in[shc+shp-1] = tex2D(tex1_32f_C1__, xx, yy+1.0f); // left-lower corner (block)
      }
      else if (tx == 1) // at left block border (inside image w.r.t x)
      {
        if (y == 0)
          sh_in[shc-shp-1] = tex2D(tex1_32f_C1__, xx-1.0f, yy); // left-upper corner (block, outside)
        else if (ty == 1)
          sh_in[shc-shp-1] = tex2D(tex1_32f_C1__, xx-1.0f, yy-1.0f); // left-upper corner (block, inside)

        sh_in[shc-1] = tex2D(tex1_32f_C1__, xx-1.0f, yy); // left border (block)

        if (y == height-1)
          sh_in[shc+shp-1] = tex2D(tex1_32f_C1__, xx-1.0f, yy); // left-lower corner (block, outside)
        else if (ty == blockDim.y)
          sh_in[shc+shp-1] = tex2D(tex1_32f_C1__, xx-1.0f, yy+1.0f); // left-lower corner (block, inside)
      }


      if (x == width-1) // at right image border
      {
        if (y == 0)
          sh_in[shc-shp+1] = FLT_MAX; // right-upper corner (image)
        else if (ty == 1)
          sh_in[shc-shp+1] = tex2D(tex1_32f_C1__, xx, yy-1.0f); // right-upper corner (block)

        sh_in[shc+1] = sh_in[shc]; // right border (image)

        if (y == height-1)
          sh_in[shc+shp+1] = FLT_MAX; // right-lower corner (image)
        else if (ty == blockDim.y)
          sh_in[shc+shp+1] = tex2D(tex1_32f_C1__, xx, yy+1.0f); // right-lower corner (block)
      }
      else if (tx == blockDim.x) // at right block border (inside image w.r.t x)
      {
        if (y == 0)
          sh_in[shc-shp+1] = tex2D(tex1_32f_C1__, xx+1.0f, yy); // right-upper corner (block, outside)
        else if (ty == 1)
          sh_in[shc-shp+1] = tex2D(tex1_32f_C1__, xx+1.0f, yy-1.0f); // right-upper corner (block, inside)

        sh_in[shc+1] = tex2D(tex1_32f_C1__, xx+1.0f, yy); // right border (block)

        if (y == height-1)
          sh_in[shc+shp+1] = tex2D(tex1_32f_C1__, xx+1.0f, yy); // right-lower corner (block, outside)
        else if (ty == blockDim.y)
          sh_in[shc+shp+1] = tex2D(tex1_32f_C1__, xx+1.0f, yy+1.0f); // right-lower corner (block, inside)
      }

      if (y == 0)
        sh_in[shc-shp] = sh_in[shc]; // upper border (image)
      else if (ty == 1)
        sh_in[shc-shp] = tex2D(tex1_32f_C1__, xx, yy-1.0f); // upper border (block)

      if (y == height-1)
        sh_in[shc+shp] = sh_in[shc]; // lower border (image)
      else if (ty == blockDim.y)
        sh_in[shc+shp] = tex2D(tex1_32f_C1__, xx, yy+1.0f); // lower border (block)

      __syncthreads();
    }

    // in a sequence of nine elements, we have to remove four times the maximum from the sequence and need
    // a fifth calculated maximum which is the median!

    float maximum;
    {
      float vals[8];

      // first 'loop'
      vals[0] = fmin(sh_in[shc-shp-1], sh_in[shc-shp]);
      maximum = fmax(sh_in[shc-shp-1], sh_in[shc-shp]);
      vals[1] = fmin(maximum, sh_in[shc-shp+1]);
      maximum = fmax(maximum, sh_in[shc-shp+1]);
      vals[2] = fmin(maximum, sh_in[shc-1]);
      maximum = fmax(maximum, sh_in[shc-1]);
      vals[3] = fmin(maximum, sh_in[shc]);
      maximum = fmax(maximum, sh_in[shc]);
      vals[4] = fmin(maximum, sh_in[shc+1]);
      maximum = fmax(maximum, sh_in[shc+1]);
      vals[5] = fmin(maximum, sh_in[shc+shp-1]);
      maximum = fmax(maximum, sh_in[shc+shp-1]);
      vals[6] = fmin(maximum, sh_in[shc+shp]);
      maximum = fmax(maximum, sh_in[shc+shp]);
      vals[7] = fmin(maximum, sh_in[shc+shp+1]);
      maximum = fmax(maximum, sh_in[shc+shp+1]);

      // second 'loop'
      maximum = fmax(vals[0], vals[1]);
      vals[0] = fmin(vals[0], vals[1]);
      vals[1] = maximum;
      maximum = fmax(vals[1], vals[2]);
      vals[1] = fmin(vals[1], vals[2]);
      vals[2] = maximum;
      maximum = fmax(vals[2], vals[3]);
      vals[2] = fmin(vals[2], vals[3]);
      vals[3] = maximum;
      maximum = fmax(vals[3], vals[4]);
      vals[3] = fmin(vals[3], vals[4]);
      vals[4] = maximum;
      maximum = fmax(vals[4], vals[5]);
      vals[4] = fmin(vals[4], vals[5]);
      vals[5] = maximum;
      maximum = fmax(vals[5], vals[6]);
      vals[5] = fmin(vals[5], vals[6]);
      vals[6] = fmin(maximum, vals[7]);

      // third 'loop'
      maximum = fmax(vals[0], vals[1]);
      vals[0] = fmin(vals[0], vals[1]);
      vals[1] = maximum;
      maximum = fmax(vals[1], vals[2]);
      vals[1] = fmin(vals[1], vals[2]);
      vals[2] = maximum;
      maximum = fmax(vals[2], vals[3]);
      vals[2] = fmin(vals[2], vals[3]);
      vals[3] = maximum;
      maximum = fmax(vals[3], vals[4]);
      vals[3] = fmin(vals[3], vals[4]);
      vals[4] = maximum;
      maximum = fmax(vals[4], vals[5]);
      vals[4] = fmin(vals[4], vals[5]);
      vals[5] = fmin(maximum, vals[6]);

      // 4th 'loop'
      maximum = fmax(vals[0], vals[1]);
      vals[0] = fmin(vals[0], vals[1]);
      vals[1] = maximum;
      maximum = fmax(vals[1], vals[2]);
      vals[1] = fmin(vals[1], vals[2]);
      vals[2] = maximum;
      maximum = fmax(vals[2], vals[3]);
      vals[2] = fmin(vals[2], vals[3]);
      vals[3] = maximum;
      maximum = fmax(vals[3], vals[4]);
      vals[3] = fmin(vals[3], vals[4]);
      vals[4] = fmin(maximum, vals[5]);

      // 5th 'loop'
      maximum = fmax(vals[0], vals[1]);
      maximum = fmax(maximum, vals[2]);
      maximum = fmax(maximum, vals[3]);
      maximum = fmax(maximum, vals[4]);
    }
    dst[oc] = maximum;
  }
}

// ----------------------------------------------------------------------------
// wrapper: median filter; 32-bit; 1-channel
IuStatus cuFilterMedian3x3(const iu::ImageGpu_32f_C1* src, iu::ImageGpu_32f_C1* dst, const IuRect& roi)
{
  // bind textures
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
  tex1_32f_C1__.filterMode = hipFilterModeLinear;
  tex1_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C1__.normalized = false;
  hipBindTexture2D(0, &tex1_32f_C1__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x), iu::divUp(roi.height, dimBlock.y));

  size_t shared_size = (block_size+2)*(block_size+2)*sizeof(float);

  cuFilterMedian3x3Kernel_32f_C1 <<< dimGrid, dimBlock, shared_size >>> (
    dst->data(roi.x, roi.y), dst->stride(), roi.x, roi.y, roi.width, roi.height);

  // unbind textures
  hipUnbindTexture(&tex1_32f_C1__);

  // error check
  return iu::checkCudaErrorState();
}

// ----------------------------------------------------------------------------
// kernel: Gaussian filter; 32-bit; 1-channel
/** Perform a convolution with an gaussian smoothing kernel
 * @param dst          pointer to output image (linear memory)
 * @param stride       length of image row [pixels]
 * @param xoff         x-coordinate offset where to start the region [pixels]
 * @param yoff         y-coordinate offset where to start the region [pixels]
 * @param width        width of region [pixels]
 * @param height       height of region [pixels]
 * @param sigma        sigma of the smoothing kernel
 * @param kernel_size  lenght of the smoothing kernel [pixels]
 * @param horizontal   defines the direction of convolution
 */
__global__ void cuFilterGaussKernel_32f_C1(float* dst, const size_t stride,
                                           const int xoff, const int yoff,
                                           const int width, const int height,
                                           float sigma, int kernel_size, bool horizontal=true)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int oc = y*stride+x;

  x += xoff;
  y += yoff;

  float xx = x+0.5f;
  float yy = y+0.5f;

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    float sum = 0.0f;
    int half_kernel_elements = (kernel_size - 1) / 2;

    if (horizontal)
    {
      // convolve horizontally
      float g0 = 1.0f / (sqrt(2.0f * 3.141592653589793f) * sigma);
      float g1 = exp(-0.5f / (sigma * sigma));
      float g2 = g1 * g1;
      sum = g0 * tex2D(tex1_32f_C1__, xx, yy);
      float sum_coeff = g0;
      for (int i = 1; i <= half_kernel_elements; i++)
      {
        g0 *= g1;
        g1 *= g2;
        float cur_xx = IUMAX(0.5f, IUMIN(width-0.5f, xx + i));
        sum += g0 * tex2D(tex1_32f_C1__, cur_xx, yy);
        cur_xx = IUMAX(0.5f, IUMIN(width-0.5f, xx-i));
        sum += g0 * tex2D(tex1_32f_C1__, cur_xx, yy);
        sum_coeff += 2.0f*g0;
      }
      dst[oc] = sum/sum_coeff;
    }
    else
    {
      // convolve vertically
      float g0 = 1.0f / (sqrt(2.0f * 3.141592653589793f) * sigma);
      float g1 = exp(-0.5f / (sigma * sigma));
      float g2 = g1 * g1;
      sum = g0 * tex2D(tex1_32f_C1__, xx, yy);
      float sum_coeff = g0;
      for (int j = 1; j <= half_kernel_elements; j++)
      {
        g0 *= g1;
        g1 *= g2;
        float cur_yy = IUMAX(0.5f, IUMIN(height-0.5f, yy+j));
        sum += g0 * tex2D(tex1_32f_C1__, xx, cur_yy);
        cur_yy = IUMAX(0.5f, IUMIN(height-0.5f, yy-j));
        sum += g0 *  tex2D(tex1_32f_C1__, xx, cur_yy);
        sum_coeff += 2.0f*g0;
      }
      dst[oc] = sum/sum_coeff;
    }
  }
}

// ----------------------------------------------------------------------------
// kernel: Gaussian filter; 32-bit; 4-channel
/** Perform a convolution with an gaussian smoothing kernel
 * @param dst          pointer to output image (linear memory)
 * @param stride       length of image row [pixels]
 * @param xoff         x-coordinate offset where to start the region [pixels]
 * @param yoff         y-coordinate offset where to start the region [pixels]
 * @param width        width of region [pixels]
 * @param height       height of region [pixels]
 * @param sigma        sigma of the smoothing kernel
 * @param kernel_size  lenght of the smoothing kernel [pixels]
 * @param horizontal   defines the direction of convolution
 */
__global__ void cuFilterGaussKernel_32f_C4(float4* dst, const size_t stride,
                                           const int xoff, const int yoff,
                                           const int width, const int height,
                                           float sigma, int kernel_size, bool horizontal=true)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int oc = y*stride+x;

  x += xoff;
  y += yoff;

  float xx = x+0.5f;
  float yy = y+0.5f;

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    int half_kernel_elements = (kernel_size - 1) / 2;

    if (horizontal)
    {
      // convolve horizontally
      float g0 = 1.0f / (sqrt(2.0f * 3.141592653589793f) * sigma);
      float g1 = exp(-0.5f / (sigma * sigma));
      float g2 = g1 * g1;
      sum = g0 * tex2D(tex1_32f_C4__, xx, yy);
      float sum_coeff = g0;
      for (int i = 1; i <= half_kernel_elements; i++)
      {
        g0 *= g1;
        g1 *= g2;
        float cur_xx = IUMAX(0.5f, IUMIN(width-0.5f, xx + i));
        sum += g0 * tex2D(tex1_32f_C4__, cur_xx, yy);
        cur_xx = IUMAX(0.5f, IUMIN(width-0.5f, xx-i));
        sum += g0 * tex2D(tex1_32f_C4__, cur_xx, yy);
        sum_coeff += 2.0f*g0;
      }
      dst[oc] = sum/sum_coeff;
    }
    else
    {
      // convolve vertically
      float g0 = 1.0f / (sqrt(2.0f * 3.141592653589793f) * sigma);
      float g1 = exp(-0.5f / (sigma * sigma));
      float g2 = g1 * g1;
      sum = g0 * tex2D(tex1_32f_C4__, xx, yy);
      float sum_coeff = g0;
      for (int j = 1; j <= half_kernel_elements; j++)
      {
        g0 *= g1;
        g1 *= g2;
        float cur_yy = IUMAX(0.5f, IUMIN(height-0.5f, yy+j));
        sum += g0 * tex2D(tex1_32f_C4__, xx, cur_yy);
        cur_yy = IUMAX(0.5f, IUMIN(height-0.5f, yy-j));
        sum += g0 *  tex2D(tex1_32f_C4__, xx, cur_yy);
        sum_coeff += 2.0f*g0;
      }
      dst[oc] = sum/sum_coeff;
    }
  }
}


// ----------------------------------------------------------------------------
// wrapper: Gaussian filter; 32-bit; 1-channel
IuStatus cuFilterGauss(const iu::ImageGpu_32f_C1* src, iu::ImageGpu_32f_C1* dst, const IuRect& roi, float sigma, int kernel_size)
{
  if (kernel_size == 0)
    kernel_size = max(5, (unsigned int)ceil(sigma*  3)*  2 + 1);
  if (kernel_size%2 == 0)
    ++kernel_size;

  // temporary variable for filtering (separabed kernel!)
  iu::ImageGpu_32f_C1 tmp(src->size());

  // textures
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
  tex1_32f_C1__.filterMode = hipFilterModeLinear;
  tex1_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C1__.normalized = false;

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x), iu::divUp(roi.height, dimBlock.y));

  // Convolve horizontally
  hipBindTexture2D(0, &tex1_32f_C1__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());
  cuFilterGaussKernel_32f_C1 <<< dimGrid, dimBlock >>> (tmp.data(roi.x, roi.y), tmp.stride(),
                                                        roi.x, roi.y, tmp.width(), tmp.height(),
                                                        sigma, kernel_size, false);

  // Convolve vertically
  hipBindTexture2D(0, &tex1_32f_C1__, tmp.data(), &channel_desc, tmp.width(), tmp.height(), tmp.pitch());
  cuFilterGaussKernel_32f_C1 <<< dimGrid, dimBlock >>> (dst->data(roi.x, roi.y), dst->stride(),
                                                        roi.x, roi.y, dst->width(), dst->height(),
                                                        sigma, kernel_size, true);

  // unbind textures
  hipUnbindTexture(&tex1_32f_C1__);

  // error check
  return iu::checkCudaErrorState();
}

// ----------------------------------------------------------------------------
// wrapper: Gaussian filter; 32-bit; 4-channel
IuStatus cuFilterGauss(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst, const IuRect& roi, float sigma, int kernel_size)
{
  if (kernel_size == 0)
    kernel_size = max(5, (unsigned int)ceil(sigma*  3)*  2 + 1);
  if (kernel_size%2 == 0)
    ++kernel_size;

  // temporary variable for filtering (separabed kernel!)
  iu::ImageGpu_32f_C4 tmp(src->size());

  // textures
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
  tex1_32f_C4__.filterMode = hipFilterModeLinear;
  tex1_32f_C4__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C4__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C4__.normalized = false;

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x), iu::divUp(roi.height, dimBlock.y));

  // Convolve horizontally
  hipBindTexture2D(0, &tex1_32f_C4__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());
  cuFilterGaussKernel_32f_C4 <<< dimGrid, dimBlock >>> (tmp.data(roi.x, roi.y), tmp.stride(),
                                                        roi.x, roi.y, tmp.width(), tmp.height(),
                                                        sigma, kernel_size, false);
  hipUnbindTexture(tex1_32f_C4__);

  // Convolve vertically
  hipBindTexture2D(0, &tex1_32f_C4__, tmp.data(), &channel_desc, tmp.width(), tmp.height(), tmp.pitch());
  cuFilterGaussKernel_32f_C4 <<< dimGrid, dimBlock >>> (dst->data(roi.x, roi.y), dst->stride(),
                                                        roi.x, roi.y, dst->width(), dst->height(),
                                                        sigma, kernel_size, true);
  hipUnbindTexture(&tex1_32f_C4__);

  // error check
  return iu::checkCudaErrorState();
}

//-----------------------------------------------------------------------------
// wrapper: cubic bspline coefficients prefilter.
IuStatus cuCubicBSplinePrefilter_32f_C1I(iu::ImageGpu_32f_C1 *input)
{
  const unsigned int block_size = 64;
  const unsigned int width  = input->width();
  const unsigned int height = input->height();

  dim3 dimBlockX(block_size,1,1);
  dim3 dimGridX(iu::divUp(height, block_size),1,1);
  cuSamplesToCoefficients2DX<float> <<< dimGridX, dimBlockX >>> (
    input->data(), width, height, input->stride());

  dim3 dimBlockY(block_size,1,1);
  dim3 dimGridY(iu::divUp(width, block_size),1,1);
  cuSamplesToCoefficients2DY<float> <<< dimGridY, dimBlockY >>> (
    input->data(), width, height, input->stride());

  return iu::checkCudaErrorState();
}


// ----------------------------------------------------------------------------
// kernel: edge filter; 32-bit; 1-channel
__global__ void  cuFilterEdgeKernel_32f_C1(float2* dst, const size_t stride,
                                           const int xoff, const int yoff,
                                           const int width, const int height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  x += xoff;
  y += yoff;

  float xx = x+0.5f;
  float yy = y+0.5f;

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    dst[y*stride+x] = make_float2(tex2D(tex1_32f_C1__, xx+1.0f, yy) - tex2D(tex1_32f_C1__, xx, yy),
                                  tex2D(tex1_32f_C1__, xx, yy) - tex2D(tex1_32f_C1__, xx, yy+1.0f) );
  }
}

// ----------------------------------------------------------------------------
// wrapper: edge filter
IuStatus cuFilterEdge(const iu::ImageGpu_32f_C1* src, iu::ImageGpu_32f_C2* dst, const IuRect& roi)
{
  // bind textures
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
  tex1_32f_C1__.filterMode = hipFilterModeLinear;
  tex1_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C1__.normalized = false;
  hipBindTexture2D(0, &tex1_32f_C1__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x), iu::divUp(roi.height, dimBlock.y));

  cuFilterEdgeKernel_32f_C1 <<< dimGrid, dimBlock >>> (
    dst->data(roi.x, roi.y), dst->stride(), roi.x, roi.y, roi.width, roi.height);

  // unbind textures
  hipUnbindTexture(&tex1_32f_C1__);

  // error check
  return iu::checkCudaErrorState();
}


// ----------------------------------------------------------------------------
// kernel: edge filter + evaluation; 32-bit; 1-channel
__global__ void  cuFilterEdgeKernel_32f_C1(float* dst, float alpha, float beta, float minval,
                                           const size_t stride,
                                           const int xoff, const int yoff,
                                           const int width, const int height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  x += xoff;
  y += yoff;

  float xx = x+0.5f;
  float yy = y+0.5f;

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    float2 grad = make_float2(tex2D(tex1_32f_C1__, xx+1.0f, yy) - tex2D(tex1_32f_C1__, xx, yy),
                              tex2D(tex1_32f_C1__, xx, yy) - tex2D(tex1_32f_C1__, xx, yy+1.0f) );
    dst[y*stride+x] = max(minval, exp(-alpha*pow(length(grad), beta)));
  }
}

// ----------------------------------------------------------------------------
// wrapper: edge filter  + evaluation
IuStatus cuFilterEdge(const iu::ImageGpu_32f_C1* src, iu::ImageGpu_32f_C1* dst, const IuRect& roi,
                      float alpha, float beta, float minval)
{
  // bind textures
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
  tex1_32f_C1__.filterMode = hipFilterModeLinear;
  tex1_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C1__.normalized = false;
  hipBindTexture2D(0, &tex1_32f_C1__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x), iu::divUp(roi.height, dimBlock.y));

  cuFilterEdgeKernel_32f_C1 <<< dimGrid, dimBlock >>> (
    dst->data(roi.x, roi.y), alpha, beta, minval, dst->stride(), roi.x, roi.y, roi.width, roi.height);

  // unbind textures
  hipUnbindTexture(&tex1_32f_C1__);

  // error check
  return iu::checkCudaErrorState();
}


// ----------------------------------------------------------------------------
// kernel: edge filter + evaluation; 32-bit; 4-channel
__global__ void  cuFilterEdgeKernel_32f_C4(float* dst, float alpha, float beta, float minval,
                                           const size_t stride,
                                           const int xoff, const int yoff,
                                           const int width, const int height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  x += xoff;
  y += yoff;

  float xx = x+0.5f;
  float yy = y+0.5f;

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    float4 gradx = tex2D(tex1_32f_C4__, xx+1.0f, yy) - tex2D(tex1_32f_C4__, xx, yy);
    float4 grady = tex2D(tex1_32f_C4__, xx, yy+1.0f) - tex2D(tex1_32f_C4__, xx, yy);
    float3 grad;
    grad.x = sqrt(gradx.x*gradx.x + grady.x*grady.x);
    grad.y = sqrt(gradx.y*gradx.y + grady.y*grady.y);
    grad.z = sqrt(gradx.z*gradx.z + grady.z*grady.z);
    dst[y*stride+x] = max(minval, exp(-alpha*pow((grad.x+grad.y+grad.z)/3.0f, beta)));
  }
}

// ----------------------------------------------------------------------------
// wrapper: edge filter  + evaluation
IuStatus cuFilterEdge(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C1* dst, const IuRect& roi,
                      float alpha, float beta, float minval)
{
  // bind textures
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
  tex1_32f_C4__.filterMode = hipFilterModeLinear;
  tex1_32f_C4__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C4__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C4__.normalized = false;
  hipBindTexture2D(0, &tex1_32f_C4__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(roi.width, dimBlock.x), iu::divUp(roi.height, dimBlock.y));

  cuFilterEdgeKernel_32f_C4 <<< dimGrid, dimBlock >>> (
    dst->data(roi.x, roi.y), alpha, beta, minval, dst->stride(), roi.x, roi.y, roi.width, roi.height);

  // unbind textures
  hipUnbindTexture(&tex1_32f_C4__);

  // error check
  return iu::checkCudaErrorState();
}

} // namespace iuprivate

#endif // IUPRIVATE_FILTER_CU

