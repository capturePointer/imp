#include <imp/cu_correspondence/solver_stereo_precond_huber_l1.cuh>

#include <hip/hip_runtime.h>

#include <imp/cu_correspondence/variational_stereo_parameters.hpp>
#include <imp/cu_core/cu_image_gpu.cuh>
#include <imp/cu_imgproc/cu_image_filter.cuh>
#include <imp/cu_imgproc/cu_resample.cuh>
#include <imp/cu_core/cu_utils.hpp>
#include <imp/cu_core/cu_texture.cuh>
#include <imp/cu_core/cu_math.cuh>

#include "warped_gradients_kernel.cuh"
#include "solver_precond_huber_l1_kernel.cuh"

namespace imp {
namespace cu {

//------------------------------------------------------------------------------
SolverStereoPrecondHuberL1::~SolverStereoPrecondHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
SolverStereoPrecondHuberL1::SolverStereoPrecondHuberL1(
    const Parameters::Ptr& params,
    imp::Size2u size,
    size_type level)
  : SolverStereoAbstract(params, size, level)
{
  u_.reset(new ImageGpu32fC1(size));
  u_prev_.reset(new ImageGpu32fC1(size));
  u0_.reset(new ImageGpu32fC1(size));
  pu_.reset(new ImageGpu32fC2(size));
  q_.reset(new ImageGpu32fC1(size));
  ix_.reset(new ImageGpu32fC1(size));
  it_.reset(new ImageGpu32fC1(size));
  xi_.reset(new ImageGpu32fC1(size));

  // and its textures
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  q_tex_ =  q_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
  xi_tex_ =  xi_->genTexture(false, hipFilterModeLinear);
}

//------------------------------------------------------------------------------
void SolverStereoPrecondHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  q_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void SolverStereoPrecondHuberL1::init(const SolverStereoAbstract& rhs)
{
  const SolverStereoPrecondHuberL1* from =
      dynamic_cast<const SolverStereoPrecondHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(*from->u0_, *from->u_);
    imp::cu::resample(*u_, *from->u0_, imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(*u_, *from->u_, imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(*pu_, *from->pu_, imp::InterpolationMode::point, false);
  imp::cu::resample(*q_, *from->q_, imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void SolverStereoPrecondHuberL1::solve(std::vector<ImageGpu32fC1::Ptr> images)
{
  if (params_->verbose > 0)
    std::cout << "StereoCtFWarpingLevelPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO

  // image textures
  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);
  u_->copyTo(*u_prev_);
  Fragmentation<16,16> frag(size_);

  // constants
  constexpr float tau = 0.95f;
  constexpr float sigma = 0.95f;
  float lin_step = 0.5f;

  // precond
  constexpr float eta = 2.0f;

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    u_->copyTo(*u0_);

    // compute warped spatial and temporal gradients
    k_warpedGradients
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (ix_->data(), it_->data(), ix_->stride(), ix_->width(), ix_->height(),
             *i1_tex_, *i2_tex_, *u0_tex_);

    // compute preconditioner
    k_preconditioner
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             params_->lambda, *ix_tex_);


    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // dual update kernel
      k_dualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(), q_->data(), q_->stride(),
               size_.width(), size_.height(),
               params_->lambda, params_->eps_u, sigma, eta,
               *u_prev_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *it_tex_);

      // and primal update kernel
      k_primalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               params_->lambda, tau, lin_step,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_);
    } // iters
    lin_step /= 1.2f;

  } // warps
  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

