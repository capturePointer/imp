#include "hip/hip_runtime.h"
#include <imp/cudepth/solver_epipolar_stereo_precond_huber_l1.cuh>

#include <hip/hip_runtime.h>

#include <glog/logging.h>

#include <imp/cudepth/variational_stereo_parameters.hpp>
#include <imp/cucore/cu_image_gpu.cuh>
#include <imp/cuimgproc/cu_image_filter.cuh>
#include <imp/cuimgproc/cu_image_transform.cuh>
#include <imp/cucore/cu_utils.hpp>
#include <imp/cucore/cu_texture.cuh>
#include <imp/cucore/cu_math.cuh>
#include <imp/cucore/cu_k_setvalue.cuh>

#include "cu_k_warped_gradients.cuh"
#include "cu_k_stereo_ctf_warping_level_precond_huber_l1.cuh"
//#include "k_epipolar_stereo_precond_huber_l1.cu"

namespace imp {
namespace cu {

//------------------------------------------------------------------------------
SolverEpipolarStereoPrecondHuberL1::~SolverEpipolarStereoPrecondHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
SolverEpipolarStereoPrecondHuberL1::SolverEpipolarStereoPrecondHuberL1(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level,
    const std::vector<cu::PinholeCamera>& cams,
    const cu::Matrix3f& F,
    const cu::SE3<float>& T_mov_fix,
    const imp::cu::ImageGpu32fC1& depth_proposal,
    const imp::cu::ImageGpu32fC1& depth_proposal_sigma2)
  : SolverStereoAbstract(params, size, level)
{
  u_.reset(new DisparityImage(size));
  u_prev_.reset(new Image(size));
  u0_.reset(new Image(size));
  pu_.reset(new VectorImage(size));
  q_.reset(new Image(size));
  iw_.reset(new Image(size));
  ix_.reset(new Image(size));
  it_.reset(new Image(size));
  xi_.reset(new Image(size));

  depth_proposal_.reset(new DisparityImage(size));
  depth_proposal_sigma2_.reset(new DisparityImage(size));

  float scale_factor = std::pow(params->ctf.scale_factor, level);

  if (depth_proposal.size() == size)
  {
    LOG(INFO) << "Copy depth proposals " << depth_proposal.size() << " to level0 "
              << depth_proposal_->size();
    depth_proposal.copyTo(*depth_proposal_);
    depth_proposal_sigma2.copyTo(*depth_proposal_sigma2_);
  }
  else
  {
    float downscale_factor = 0.5f*((float)size.width()/(float)depth_proposal.width()+
                                   (float)size.height()/(float)depth_proposal.height());

    LOG(INFO) << "depth proposal downscaled to level: " << level << "; size: " << size
              << "; downscale_factor: " << downscale_factor;

    imp::cu::resample(*depth_proposal_, depth_proposal);
    imp::cu::resample(*depth_proposal_sigma2_, depth_proposal_sigma2);
//    *depth_proposal_ *= downscale_factor;
    //*depth_proposal_sigma2_ *= downscale_factor; //!< @todo (MWE) do we need to scale this?
  }

  F_ = F;
  T_mov_fix_ = T_mov_fix;

  // assuming we receive the camera matrix for level0
  if  (level == 0)
  {
    cams_ = cams;
  }
  else
  {
    for (auto cam : cams)
    {
      cu::PinholeCamera scaled_cam = cam * scale_factor;
      cams_.push_back(scaled_cam);
    }
  }

//  imp::Pixel32fC1 min_val, max_val;
//  imp::cu::minMax(*depth_proposal_tex_, min_val, max_val, size);
//  LOG(INFO) << "depth_proposal_tex_: " << min_val << " - " << max_val;
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  q_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::init(const SolverStereoAbstract& rhs)
{
  const SolverEpipolarStereoPrecondHuberL1* from =
      dynamic_cast<const SolverEpipolarStereoPrecondHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(*from->u0_, *from->u_);
    imp::cu::resample(*u_, *from->u0_, imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(*u_, *from->u_, imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(*pu_, *from->pu_, imp::InterpolationMode::point, false);
  imp::cu::resample(*q_, *from->q_, imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::solve(std::vector<ImagePtr> images)
{
  if (params_->verbose > 0)
    std::cout << "SolverEpipolarStereoPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO

  // textures
  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  q_tex_ =  q_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
  xi_tex_ =  xi_->genTexture(false, hipFilterModeLinear);
  depth_proposal_tex_ =  depth_proposal_->genTexture(false, hipFilterModeLinear);
  depth_proposal_sigma2_tex_ =  depth_proposal_sigma2_->genTexture(false, hipFilterModeLinear);


  u_->copyTo(*u_prev_);
  Fragmentation<16,16> frag(size_);

  // constants
  constexpr float tau = 0.95f;
  constexpr float sigma = 0.95f;
  float lin_step = 0.5f;

  // precond
  constexpr float eta = 2.0f;

  std::cout << "F: " << F_ << std::endl;

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    u_->copyTo(*u0_);


    // compute warped spatial and temporal gradients
    k_warpedGradientsEpipolarConstraint
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (iw_->data(), ix_->data(), it_->data(), ix_->stride(), ix_->width(), ix_->height(),
             cams_.at(0), cams_.at(1), F_, T_mov_fix_,
             *i1_tex_, *i2_tex_, *u0_tex_,
             *depth_proposal_tex_);

    // compute preconditioner
    k_preconditioner
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             params_->lambda, *ix_tex_);


    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // dual update kernel
      k_dualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(), q_->data(), q_->stride(),
               size_.width(), size_.height(),
               params_->lambda, params_->eps_u, sigma, eta,
               *u_prev_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *it_tex_);

      // and primal update kernel
      k_primalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               params_->lambda, tau, lin_step,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_);
    } // iters
    lin_step /= 1.2f;

  } // warps



  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

