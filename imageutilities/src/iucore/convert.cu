#include "hip/hip_runtime.h"
/*
 * Copyright (c) ICG. All rights reserved.
 *
 * Institute for Computer Graphics and Vision
 * Graz University of Technology / Austria
 *
 *
 * This software is distributed WITHOUT ANY WARRANTY; without even
 * the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE.  See the above copyright notices for more information.
 *
 *
 * Project     : ImageUtilities
 * Module      : Core
 * Class       : none
 * Language    : C/CUDA
 * Description : CUDA kernels for core functions
 *
 * Author     : Manuel Werlberger
 * EMail      : werlberger@icg.tugraz.at
 *
 */

#ifndef IUCORE_CONVERT_CU
#define IUCORE_CONVERT_CU

#include <cutil_math.h>
#include "coredefs.h"
#include "memorydefs.h"
#include "iutextures.cuh"

namespace iuprivate {


/* ***************************************************************************
 *  CUDA KERNELS
 * ***************************************************************************/

//-----------------------------------------------------------------------------
/** convert kernel 32f_C3 -> 32f_C4 (float3 -> float4)
 */
__global__ void cuConvertC3ToC4Kernel(const float3* src, size_t src_stride, int src_width, int src_height,
                                      float4* dst, size_t dst_stride, int dst_width, int dst_height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    float3 val=src[src_c];
    dst[dst_c] =  make_float4(val.x, val.y, val.z, 1.0f);
  }
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 32f_C3 (float4 -> float3)
 */
__global__ void cuConvertC4ToC3Kernel(const float4* src, size_t src_stride, int src_width, int src_height,
                                      float3* dst, size_t dst_stride, int dst_width, int dst_height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    float4 val=src[src_c];
    dst[dst_c] = make_float3(val.x, val.y, val.z);
  }
}


//-----------------------------------------------------------------------------
/** convert kernel 8u_C1 -> 32f_C1 (unsigned char -> float)
 */
__global__ void cuConvert8uC1To32fC1Kernel(const unsigned char *src, size_t src_stride, int src_width, int src_height,
                                      float* dst, size_t dst_stride, int dst_width, int dst_height, float mul_constant,
                                       float add_constant)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C1 -> 8u_C1 (float -> unsigned char)
 */
__global__ void cuConvert32fC1To8uC1Kernel(const float* src, size_t src_stride, int src_width, int src_height,
                                      unsigned char* dst, size_t dst_stride, int dst_width, int dst_height, float mul_constant,
                                       unsigned char add_constant)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int src_c = y*src_stride + x;
  int dst_c = y*dst_stride + x;

  if (x<src_width && y<src_height && x<dst_width && y<dst_height)
  {
    dst[dst_c] = src[src_c] * mul_constant + add_constant;
  }
}

//-----------------------------------------------------------------------------
/** convert kernel rgb -> hsv
 */
__global__ void cuConvertRGBToHSVKernel(const float4* src, float4* dst, size_t stride, int width, int height,
                                        bool normalize)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];
    float M = IUMAX(in.x, IUMAX(in.y, in.z));
    float m = IUMIN(in.x, IUMIN(in.y, in.z));
    float C = M-m;

    // Hue
    float H = 0.0f;
    if (M == in.x)
      H = fmod((in.y - in.z)/C, 6.0f);
    if (M == in.y)
      H = (in.z - in.x)/C + 2.0f;
    if (M == in.z)
      H = (in.x - in.y)/C + 4.0f;
    H = H*60.0f;

    // Value
    float V = M;

    // Saturation
    float S = 0.0f;
    if (C != 0.0f)
      S = C/V;

    // Normalize
    if (normalize)
      H = H/360.0f;

    // Write Back
    dst[c] = make_float4(H, S, V, in.w);
  }
}

//-----------------------------------------------------------------------------
/** convert kernel hsv -> rgb
 */
__global__ void cuConvertHSVToRGBKernel(const float4* src, float4* dst, size_t stride, int width, int height,
                                        bool denormalize)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  int c = y*stride + x;

  if (x<width && y<height)
  {
    // Read
    float4 in = src[c];

    float C = in.z*in.y;

    // Denormalize
    if (denormalize)
      in.x = in.x*360.0f;

    // RGB
    float H = in.x/60.0f;
    float X = C*(1.0f - abs(fmod(H, 2.0f) - 1.0f));

    float4 rgb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    if (H >= 0.0f)
    {
      if (H < 1.0f)
        rgb = make_float4(C, X, 0.0f, 0.0f);
      else if (H < 2.0f)
        rgb = make_float4(X, C, 0.0f, 0.0f);
      else if (H < 3.0f)
        rgb = make_float4(0.0f, C, X, 0.0f);
      else if (H < 4.0f)
        rgb = make_float4(0.0f, X, C, 0.0f);
      else if (H < 5.0f)
        rgb = make_float4(X, 0.0f, C, 0.0f);
      else if (H < 6.0f)
        rgb = make_float4(C, 0.0f, X, 0.0f);
    }

    float m = in.z-C;
    rgb += m;

    // Write Back
    rgb.w = in.w;
    dst[c] = rgb;
  }
}


/* ***************************************************************************
 *  CUDA WRAPPERS
 * ***************************************************************************/

//-----------------------------------------------------------------------------
/** convert kernel 32f_C3 -> 32f_C4 (float3 -> float4)
 */
IuStatus cuConvert(const iu::ImageGpu_32f_C3* src, const IuRect& src_roi,
                   iu::ImageGpu_32f_C4* dst, const IuRect& dst_roi)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvertC3ToC4Kernel <<< dimGrid, dimBlock >>> (
      src->data(src_roi.x, src_roi.y), src->stride(), src_roi.width, src_roi.height,
      dst->data(dst_roi.x, dst_roi.y), dst->stride(), dst_roi.width, dst_roi.height);

  IU_CHECK_AND_RETURN_CUDA_ERRORS();
}

//-----------------------------------------------------------------------------
/** convert kernel 32f_C4 -> 32f_C3 (float4 -> float3)
 */
IuStatus cuConvert(const iu::ImageGpu_32f_C4* src, const IuRect& src_roi,
                   iu::ImageGpu_32f_C3* dst, const IuRect& dst_roi)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvertC4ToC3Kernel <<< dimGrid, dimBlock >>> (
      src->data(src_roi.x, src_roi.y), src->stride(), src_roi.width, src_roi.height,
      dst->data(dst_roi.x, dst_roi.y), dst->stride(), dst_roi.width, dst_roi.height);

  IU_CHECK_AND_RETURN_CUDA_ERRORS();
}


//-----------------------------------------------------------------------------
IuStatus cuConvert_8u_32f(const iu::ImageGpu_8u_C1* src, const IuRect& src_roi,
                   iu::ImageGpu_32f_C1* dst, const IuRect& dst_roi, float mul_constant,
                   float add_constant)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvert8uC1To32fC1Kernel <<< dimGrid, dimBlock >>> (
      src->data(src_roi.x, src_roi.y), src->stride(), src_roi.width, src_roi.height,
      dst->data(dst_roi.x, dst_roi.y), dst->stride(), dst_roi.width, dst_roi.height, mul_constant, add_constant);

  IU_CHECK_AND_RETURN_CUDA_ERRORS();
}


//-----------------------------------------------------------------------------
IuStatus cuConvert_32f_8u(const iu::ImageGpu_32f_C1* src, const IuRect& src_roi,
                   iu::ImageGpu_8u_C1* dst, const IuRect& dst_roi, float mul_constant,
                   unsigned char add_constant)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(dst_roi.width - dst_roi.x, dimBlock.x),
               iu::divUp(dst_roi.height - dst_roi.y, dimBlock.y));

  cuConvert32fC1To8uC1Kernel <<< dimGrid, dimBlock >>> (
      src->data(src_roi.x, src_roi.y), src->stride(), src_roi.width, src_roi.height,
      dst->data(dst_roi.x, dst_roi.y), dst->stride(), dst_roi.width, dst_roi.height, mul_constant, add_constant);

  IU_CHECK_AND_RETURN_CUDA_ERRORS();
}

//-----------------------------------------------------------------------------
IuStatus cuConvert_rgb_to_hsv(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst, bool normalize)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertRGBToHSVKernel <<< dimGrid, dimBlock >>> (
      src->data(), dst->data(), src->stride(), src->width(), src->height(), normalize);

  IU_CHECK_AND_RETURN_CUDA_ERRORS();
}

//-----------------------------------------------------------------------------
IuStatus cuConvert_hsv_to_rgb(const iu::ImageGpu_32f_C4* src, iu::ImageGpu_32f_C4* dst, bool denormalize)
{
  // fragmentation
  const unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(iu::divUp(src->width(), dimBlock.x),
               iu::divUp(src->height(), dimBlock.y));

  cuConvertHSVToRGBKernel <<< dimGrid, dimBlock >>> (
      src->data(), dst->data(), src->stride(), src->width(), src->height(), denormalize);

  IU_CHECK_AND_RETURN_CUDA_ERRORS();
}



} // namespace iuprivate

#endif // IUCORE_CONVERT_CU
