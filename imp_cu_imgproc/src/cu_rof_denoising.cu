#include "hip/hip_runtime.h"
#include <imp/cu_imgproc/cu_rof_denoising.cuh>

#include <iostream>

#include <hip/hip_runtime.h>

#include <imp/cuda_toolkit/hip/hip_vector_types.h>
#include <imp/core/pixel.hpp>
#include <imp/cu_core/cu_texture.cuh>
#include <imp/cu_core/cu_k_derivative.cuh>
#include <imp/cu_core/cu_math.cuh>


namespace imp {
namespace cu {

//-----------------------------------------------------------------------------
__global__ void k_initRofSolver(Pixel32fC1* d_u, Pixel32fC1* d_u_prev, size_t stride_u,
                                Pixel32fC2* d_p, size_t stride_p,
                                imp::cu::Texture2D f_tex,
                                size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float val = tex2DFetch<float>(f_tex, x, y);
    d_u[y*stride_u + x] = val;
    d_u_prev[y*stride_u + x] = val;
    d_p[y*stride_p + x] = Pixel32fC2(0.0f, 0.0f);
  }
}

//-----------------------------------------------------------------------------
__global__ void k_rofPrimalUpdate(
    Pixel32fC1* d_u, Pixel32fC1* d_u_prev, size_t stride_u,
    Texture2D f_tex, Texture2D u_tex, Texture2D p_tex,
    float lambda, float tau, float theta, size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float f = tex2DFetch<float>(f_tex, x, y);
    float u = tex2DFetch<float>(u_tex, x, y);
    float u_prev = u;
    float div = dpAd(p_tex, x, y, width, height);

    u = (u + tau*(div + lambda*f)) / (1.0f + tau*lambda);

    d_u[y*stride_u + x] = u;
    d_u_prev[y*stride_u + x] = u + theta*(u-u_prev);
  }
}

//-----------------------------------------------------------------------------
__global__ void k_rofDualUpdate(
    Pixel32fC2* d_p, size_t stride_p, Texture2D p_tex, Texture2D u_prev_tex,
    float sigma, size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float2 p = tex2DFetch<float2>(p_tex, x, y);
    float2 dp_u = dp(u_prev_tex, x, y);

    p = p + sigma*dp_u;
    p = p / max(1.0f, length(p));
    d_p[y*stride_p + x] = {p.x, p.y};
  }
}

//-----------------------------------------------------------------------------
__global__ void k_convertResult8uC1(Pixel8uC1* d_u, size_t stride_u,
                                    imp::cu::Texture2D u_tex,
                                    size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    d_u[y*stride_u + x] = static_cast<std::uint8_t>(
          255.0f * tex2DFetch<float>(u_tex, x, y));
  }
}

//-----------------------------------------------------------------------------
__global__ void k_rofPrimalEnergy(Pixel32fC1* d_ep,  size_type stride,
                                  std::uint32_t width, std::uint32_t height,
                                  float lambda, Texture2D f_tex, Texture2D u_tex)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float2 dp_u = dp(u_tex, x, y);
    float f = tex2DFetch<float>(f_tex, x, y);
    float u = tex2DFetch<float>(u_tex, x, y);
    d_ep[y*stride + x] = length(dp_u) + lambda/2.0f * imp::cu::sqr(u-f);
  }
}

//-----------------------------------------------------------------------------
__global__ void k_rofDualEnergy(Pixel32fC1* d_ed,  size_type stride,
                                std::uint32_t width, std::uint32_t height,
                                float lambda, Texture2D f_tex, Texture2D p_tex)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float f = tex2DFetch<float>(f_tex, x, y);
    float div = dpAd(p_tex, x, y, width, height);
    d_ed[y*stride + x] = -imp::cu::sqr(div)/(2.0f*lambda) - div*f;
  }
}



//#############################################################################

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void RofDenoising<Pixel, pixel_type>::init(const Size2u& size)
{
  Base::init(size);
  IMP_CUDA_CHECK();

  // setup textures
  f_tex_ = f_->genTexture(false, hipFilterModeLinear, hipAddressModeClamp,
                          (f_->bitDepth()==8) ? hipReadModeNormalizedFloat :
                                                hipReadModeElementType);
  u_tex_ = u_->genTexture(false, hipFilterModeLinear, hipAddressModeClamp,
                          hipReadModeElementType);
  u_prev_tex_ = u_prev_->genTexture(false, hipFilterModeLinear,
                                    hipAddressModeClamp, hipReadModeElementType);
  p_tex_ = p_->genTexture(false, hipFilterModeLinear, hipAddressModeClamp,
                          hipReadModeElementType);
  IMP_CUDA_CHECK();

  // init internal vars
  k_initRofSolver
      <<< dimGrid(), dimBlock() >>> (u_->data(), u_prev_->data(), u_->stride(),
                                     p_->data(), p_->stride(),
                                     *f_tex_, size_.width(), size_.height());
  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void RofDenoising<Pixel, pixel_type>::denoise(const std::shared_ptr<ImageBase>& dst,
                                              const std::shared_ptr<ImageBase>& src)
{
  if (params_.verbose)
  {
    std::cout << "[Solver @gpu] RofDenoising::denoise:" << std::endl;
  }

  if (src->size() != dst->size())
  {
    throw imp::cu::Exception("Input and output image are not of the same size.",
                             __FILE__, __FUNCTION__, __LINE__);
  }

  f_ = std::dynamic_pointer_cast<ImageGpu>(src);
  //! @todo (MWE) we could use dst for u_ if pixel_type is consistent

  if (size_ != f_->size())
  {
    this->init(f_->size());
  }

  // internal params
  float L = sqrtf(8.0f);
  float tau = 1/L;
  float sigma = 1/L;
  float theta = 1.0f;

  for(int iter = 0; iter < this->params_.max_iter; ++iter)
  {
    if (sigma < 1000.0f)
      theta = 1.f/sqrtf(1.0f+0.7f*this->params_.lambda*tau);
    else
      theta = 1.0f;

    if (params_.verbose)
    {
      std::cout << "(rof solver) iter: " << iter << "; tau: " << tau
                << "; sigma: " << sigma << "; theta: " << theta << std::endl;
    }

    if (params_.primal_dual_energy_check_iter > 0
        && iter % params_.primal_dual_energy_check_iter
        && params_.primal_dual_gap_tolerance > 0.f)
    {
      double primal_energy = 0.0, dual_energy = 0.0;
      this->primalDualEnergy(primal_energy, dual_energy);
      std::cout << "ENERGIES: primal: " << primal_energy <<
                   "; dual: " << dual_energy << std::endl;
    }

    k_rofDualUpdate
        <<< dimGrid(), dimBlock() >>> (p_->data(), p_->stride(),
                                       *p_tex_, *u_prev_tex_,
                                       sigma, size_.width(), size_.height());

    k_rofPrimalUpdate
        <<< dimGrid(), dimBlock() >>> (u_->data(), u_prev_->data(), u_->stride(),
                                       *f_tex_, *u_tex_, *p_tex_,
                                       params_.lambda, tau, theta,
                                       size_.width(), size_.height());

    sigma /= theta;
    tau *= theta;
  }
  IMP_CUDA_CHECK();

  switch (dst->pixelType())
  {
  case PixelType::i8uC1:
  {
    std::shared_ptr<ImageGpu8uC1> u(std::dynamic_pointer_cast<ImageGpu8uC1>(dst));
    k_convertResult8uC1
        <<< dimGrid(), dimBlock() >>> (u->data(), u->stride(),
                                       *u_tex_, size_.width(), size_.height());
  }
  break;
  case PixelType::i32fC1:
  {
    std::shared_ptr<ImageGpu32fC1> u(std::dynamic_pointer_cast<ImageGpu32fC1>(dst));
    u_->copyTo(*u);
  }
  break;
  default:
    throw imp::cu::Exception("Unsupported PixelType.",
                             __FILE__, __FUNCTION__, __LINE__);
  }
  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void RofDenoising<Pixel, pixel_type>::primalDualEnergy(
    double& primal_energy, double& dual_energy)
{
  if (!primal_energies_ || !dual_energies_)
  {
    primal_energies_.reset(new ImageGpu32fC1(size_));
    dual_energies_.reset(new ImageGpu32fC1(size_));
  }

  Pixel32fC1 ep_min, ep_max, ed_min, ed_max;

  k_rofPrimalEnergy
      <<<
         this->dimGrid(), this->dimBlock()
      >>> (primal_energies_->data(), primal_energies_->stride(),
           size_.width(), size_.height(), params_.lambda,
           *f_tex_, *u_tex_);

  // TODO sum
  primal_energy = 10.0;
  //imp::cu::minMax(*primal_energies_, ep_min, ep_max);
  IMP_CUDA_CHECK();

  k_rofDualEnergy
      <<<
         this->dimGrid(), this->dimBlock()
      >>> (dual_energies_->data(), dual_energies_->stride(),
           size_.width(), size_.height(), params_.lambda,
           *f_tex_, *p_tex_);
  dual_energy = 20.0;
  //imp::cu::minMax(*dual_energies_, ed_min, ed_max);

  std::cout << "!!! primal: min: " << ep_min << "; max: " << ep_max << std::endl;
  std::cout << "!!! dual  : min: " << ed_min << "; max: " << ed_max << std::endl;

  IMP_CUDA_CHECK();
}

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void RofDenoising<Pixel, pixel_type>::print(std::ostream& os) const
{
  os << "ROF Denoising:" << std::endl;
  this->Base::print(os);
}

//=============================================================================
// Explicitely instantiate the desired classes
// (sync with typedefs at the end of the hpp file)
template class RofDenoising<imp::Pixel8uC1, imp::PixelType::i8uC1>;
template class RofDenoising<imp::Pixel32fC1, imp::PixelType::i32fC1>;

} // namespace cu
              } // namespace imp
