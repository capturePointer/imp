#include "hip/hip_runtime.h"
/*
 * Copyright (c) ICG. All rights reserved.
 *
 * Institute for Computer Graphics and Vision
 * Graz University of Technology / Austria
 *
 *
 * This software is distributed WITHOUT ANY WARRANTY; without even
 * the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE.  See the above copyright notices for more information.
 *
 *
 * Project     : ImageUtilities
 * Module      : Geometric Transformation
 * Class       : none
 * Language    : CUDA
 * Description : Implementation of CUDA wrappers for remap operations
 *
 * Author     : Manuel Werlberger
 * EMail      : werlberger@icg.tugraz.at
 *
 */

#include <iostream>
#include <iudefs.h>
#include <iucutil.h>
#include <iucore/iutextures.cuh>

#ifndef IUTRANSFORM_REMAP_CU
#define IUTRANSFORM_REMAP_CU

namespace iuprivate {

// local textures
texture<float, 2, hipReadModeElementType> tex_remap_dx_32f_C1__;
texture<float, 2, hipReadModeElementType> tex_remap_dy_32f_C1__;

/** Remap input image (tex1) with disparities (tex_remap_dx, tex_remap_dy). */
// linear interpolation
__global__ void cuRemapKernel_32f_C1(float *dst, size_t stride, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // texutre coordinates
  const float xx = x+0.5f;
  const float yy = y+0.5f;
  // warped texutre coordinates
  const float wx = xx + tex2D(tex_remap_dx_32f_C1__, xx, yy);
  const float wy = yy + tex2D(tex_remap_dy_32f_C1__, xx, yy);

  if (x<width && y<height) // Check if out coordinates lie inside output image
  {
    dst[y*stride+x] = tex2D(tex1_32f_C1__, wx, wy);
  }
}
// cubic interpolation
__global__ void cuRemapCubicKernel_32f_C1(float *dst, size_t stride, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // texutre coordinates
  const float xx = x+0.5f;
  const float yy = y+0.5f;
  // warped texutre coordinates
  const float wx = xx + tex2D(tex_remap_dx_32f_C1__, xx, yy);
  const float wy = yy + tex2D(tex_remap_dy_32f_C1__, xx, yy);

  if (x<width && y<height) // Check if out coordinates lie inside output image
  {
    dst[y*stride+x] = iu::cubicTex2DSimple(tex1_32f_C1__, wx, wy);
  }
}
// cubic spline interpolation
__global__ void cuRemapCubicSplineKernel_32f_C1(float *dst, size_t stride, int width, int height)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // texutre coordinates
  const float xx = x+0.5f;
  const float yy = y+0.5f;
  // warped texutre coordinates
  const float wx = xx + tex2D(tex_remap_dx_32f_C1__, xx, yy);
  const float wy = yy + tex2D(tex_remap_dy_32f_C1__, xx, yy);

  if (x<width && y<height) // Check if out coordinates lie inside output image
  {
    dst[y*stride+x] = iu::cubicTex2D(tex1_32f_C1__, wx, wy);
  }
}

//-----------------------------------------------------------------------------
IuStatus cuRemap(iu::ImageGpu_32f_C1* src,
                 iu::ImageGpu_32f_C1* dx_map, iu::ImageGpu_32f_C1* dy_map,
                 iu::ImageGpu_32f_C1* dst, IuInterpolationType interpolation)
{
  tex1_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex1_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex1_32f_C1__.normalized = false;

  tex_remap_dx_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex_remap_dx_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex_remap_dx_32f_C1__.normalized = false;
  tex_remap_dx_32f_C1__.filterMode = hipFilterModePoint;

  tex_remap_dy_32f_C1__.addressMode[0] = hipAddressModeClamp;
  tex_remap_dy_32f_C1__.addressMode[1] = hipAddressModeClamp;
  tex_remap_dy_32f_C1__.normalized = false;
  tex_remap_dy_32f_C1__.filterMode = hipFilterModePoint;


  // bind src image to texture and use as input for reduction
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
  hipBindTexture2D(0, &tex1_32f_C1__, src->data(), &channel_desc, src->width(), src->height(), src->pitch());
  hipBindTexture2D(0, &tex_remap_dx_32f_C1__, dx_map->data(), &channel_desc, dx_map->width(), dx_map->height(), dx_map->pitch());
  hipBindTexture2D(0, &tex_remap_dy_32f_C1__, dy_map->data(), &channel_desc, dy_map->width(), dy_map->height(), dy_map->pitch());

  // fragmentation
  unsigned int block_size = 16;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGridOut(iu::divUp(dst->width(), dimBlock.x), iu::divUp(dst->height(), dimBlock.y));

  switch(interpolation)
  {
  case IU_INTERPOLATE_NEAREST:
  case IU_INTERPOLATE_CUBIC:
  case IU_INTERPOLATE_CUBIC_SPLINE:
    tex1_32f_C1__.filterMode = hipFilterModePoint;
    break;
  case IU_INTERPOLATE_LINEAR:
    tex1_32f_C1__.filterMode = hipFilterModeLinear;
    break;
  }

  switch(interpolation)
  {
  case IU_INTERPOLATE_NEAREST:
  case IU_INTERPOLATE_LINEAR: // fallthrough intended
    cuRemapKernel_32f_C1 <<< dimGridOut, dimBlock >>> (
        dst->data(), dst->stride(), dst->width(), dst->height());
    break;
  case IU_INTERPOLATE_CUBIC:
    cuRemapCubicKernel_32f_C1 <<< dimGridOut, dimBlock >>> (
        dst->data(), dst->stride(), dst->width(), dst->height());
    break;
  case IU_INTERPOLATE_CUBIC_SPLINE:
    cuRemapCubicSplineKernel_32f_C1 <<< dimGridOut, dimBlock >>> (
        dst->data(), dst->stride(), dst->width(), dst->height());
    break;
  }

  hipUnbindTexture(&tex1_32f_C1__);
  hipUnbindTexture(&tex_remap_dx_32f_C1__);
  hipUnbindTexture(&tex_remap_dy_32f_C1__);

  return iu::checkCudaErrorState();
}


} // namespace iuprivate

#endif // IUTRANSFORM_REMAP_CU
