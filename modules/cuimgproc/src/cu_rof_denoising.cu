#include "hip/hip_runtime.h"
#include <imp/cuimgproc/cu_rof_denoising.cuh>

#include <iostream>

#include <hip/hip_runtime.h>

#include <imp/cuda_toolkit/hip/hip_vector_types.h>
#include <imp/core/pixel.hpp>
#include <imp/cucore/cu_texture.cuh>


namespace imp { namespace cu {

//-----------------------------------------------------------------------------
/** compute forward differences in x- and y- direction */
static __device__ __forceinline__ float2 dp(
    const imp::cu::Texture2D& tex, float x, float y, size_t width, size_t height)
{
  x+=0.5f;
  y+=0.5f;
  float2 grad = make_float2(0.0f, 0.0f);
  float cval = tex2D<float>(tex, x, y);
  if (x<width-1)
  {
    grad.x = tex2D<float>(tex, x+1.f, y) - cval;
  }
  if (y<height-1)
  {
    grad.y = tex2D<float>(tex, x, y+1.f) - cval;
  }
  return grad;
}

//-----------------------------------------------------------------------------
/** compute divergence using backward differences (adjugate from dp). */
static __device__ __forceinline__
float dpAd(const imp::cu::Texture2D& tex, size_t x, size_t y, size_t width, size_t height)
{
  float2 cval = tex2D<float2>(tex, x+0.5f, y+0.5f);
  float2 wval = tex2D<float2>(tex, x-0.5f, y+0.5f);
  float2 nval = tex2D<float2>(tex, x+0.5f, y-0.5f);

  if (x == 0)
    wval.x = 0.0f;
  else if (x >= width-1)
    cval.x = 0.0f;


  if (y == 0)
    nval.y = 0.0f;
  else if (y >= height-1)
    cval.y = 0.0f;

  return (cval.x - wval.x + cval.y - nval.y);
}

//// texture object is a kernel argument
//template<typename Pixel>
//__global__ void k_simpleTextureObjectTest(Pixel* u, size_t stride_u,
//                                          imp::cu::Texture2D f_tex,
//                                          Pixel* f, size_t stride_f,
//                                          size_t width, size_t height)
//{
//  int x = blockIdx.x*blockDim.x + threadIdx.x;
//  int y = blockIdx.y*blockDim.y + threadIdx.y;

//  if (x>=0 && y>=0 && x<width && y<height)
//  {
//    float px = tex2D<float>(f_tex, x+.5f, y+.5f);
//    u[y*stride_u+x] = f[y*stride_f+x] - static_cast<int>(255.0f*px);
//  }
//}

//-----------------------------------------------------------------------------
__global__ void k_solveRofPrimalIteration(
    float* d_u, float* d_u_prev, size_t stride_8uC1,
    Texture2D f_tex, Texture2D u_tex, Texture2D p_tex,
    float lambda, float tau, float theta, size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float xx = x+0.5f;
    float yy = y+0.5f;

    float f = tex2D<float>(f_tex, xx, yy);
    float u = tex2D<float>(u_tex, xx, yy);
    float u_prev = u;
    float div = dpAd(p_tex, x, y, width, height);

    u = (u + tau*(div + lambda*f)) / (1.0f + tau*lambda);

    d_u[y*stride_8uC1 + x] = u;
    d_u_prev[y*stride_8uC1 + x] = u + theta*(u-u_prev);
  }
}

//-----------------------------------------------------------------------------
__global__ void k_solveRofDualIteration(
    float2* d_p, size_t stride_p, Texture2D p_tex, Texture2D u_prev_tex,
    float sigma, size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    float2 p = tex2D<float2>(p_tex, x+.5f, y+.5f);
    float2 dp_u = dp(u_prev_tex, x, y, width, height);

    p += sigma*dp_u;
    p /= max(1.0f, length(p));
    d_p[y*stride_p + x] = p;
  }
}

//-----------------------------------------------------------------------------
__global__ void k_convertResult8uC1(unsigned char* u, size_t stride_u,
                                    imp::cu::Texture2D u_tex,
                                    size_t width, size_t height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x<width && y<height)
  {
    u[y*stride_u + x] = static_cast<unsigned char>(
          255.0f * tex2D<float>(u_tex, x+.5f, y+.5f));
  }
}


//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void RofDenoising<Pixel, pixel_type>::RofDenoising::denoise(ImagePtr f, ImagePtr u)
{
  std::cout << "solving the ROF image denosing model (gpu)" << std::endl;

  if (f->size() != u->size())
  {
    throw imp::cu::Exception("Input and output image are not of the same size.",
                             __FILE__, __FUNCTION__, __LINE__);
  }

  this->f_ = f;

  if (this->size_ != this->f_->size()
      || this->u_prev_ == nullptr
      || this->p_ == nullptr
      || fragmentation_ == nullptr)
  {
    this->size_ = this->f_->size();
    fragmentation_.reset(new Fragmentation<16>(this->size_));

    // setup internal memory
    switch (this->u_->nChannels())
    {
    case 1:
      this->u_.reset(new ImageGpu32fC1(this->size_));
      this->u_prev_.reset(new ImageGpu32fC1(this->size_));
      this->p_.reset(new ImageGpu32fC2(this->size_));
      break;
    default:
      throw imp::cu::Exception("ROF denoising not implemented for given image type.",
                               __FILE__, __FUNCTION__, __LINE__);
    }

    // setup textures
    this->f_tex_ = this->f_->genTexture(false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp,
                                        hipReadModeNormalizedFloat);
    this->u_tex_ = this->u_->genTexture(false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp,
                                        hipReadModeElementType);
    this->u_prev_tex_ = this->u_prev_->genTexture(false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp,
                                        hipReadModeElementType);
    this->p_tex_ = this->p_->genTexture(false,
                                        hipFilterModeLinear,
                                        hipAddressModeClamp,
                                        hipReadModeElementType);

    // internal params
    float L = sqrtf(8.0f);
    float tau = 1/L;
    float sigma = 1/L;
    float theta = 1.0f;

    for(int iter = 0; iter < this->params_.max_iter; ++iter)
    {
      k_solveRofDualIteration
          <<< fragmentation_->dimGrid, fragmentation_->dimBlock >>> (
          reinterpret_cast<float2*>(this->p_->data()), this->p_->stride(),
          *this->p_tex_, *this->u_prev_tex_,
          sigma, this->size_.width(), this->size_.height());

      if (sigma < 1000.0f)
        theta = 1.f/sqrtf(1.0f+0.7f*this->params_.lambda*tau);
      else
        theta = 1.0f;

      k_solveRofPrimalIteration
          <<< fragmentation_->dimGrid, fragmentation_->dimBlock >>> (
          reinterpret_cast<float*>(this->u_->data()),
          reinterpret_cast<float*>(this->u_prev_->data()), this->u_->stride(),
          *this->f_tex_, *this->u_tex_, *this->p_tex_,
          this->params_.lambda, tau, theta, this->size_.width(), this->size_.height());

      sigma /= theta;
      theta *= theta;
    }

    // copy final result to output
//    if (u->pixelType() == this->u_->pixelType())
//    {
//      this->u_->copyTo(*u);
//    }
//    else
//    {
      switch (u->pixelType())
      {
      case PixelType::i8uC1:
        k_convertResult8uC1
            <<< fragmentation_->dimGrid, fragmentation_->dimBlock >>> (
            reinterpret_cast<unsigned char*>(u->data()), u->stride(), *this->u_tex_,
            this->size_.width(), this->size_.height());
      }

//    }


//    // call test kernel
//    k_simpleTextureObjectTest <<< fragmentation_->dimGrid, fragmentation_->dimBlock >>> (
//      this->u_->data(), this->u_->stride(), *(this->f_tex_.get()),
//      this->f_->data(), this->f_->stride(),
//      this->size_.width(), this->size_.height());

  }
}

//=============================================================================
// Explicitely instantiate the desired classes
// (sync with typedefs at the end of the hpp file)
template class RofDenoising<imp::Pixel8uC1, imp::PixelType::i8uC1>;
template class RofDenoising<imp::Pixel32fC1, imp::PixelType::i32fC1>;

} // namespace cu
} // namespace imp
