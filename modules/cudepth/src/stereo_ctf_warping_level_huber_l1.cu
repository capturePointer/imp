#include "hip/hip_runtime.h"
#include <imp/cudepth/stereo_ctf_warping_level_huber_l1.cuh>

#include <cmath>

#include <hip/hip_runtime.h>

#include <imp/cudepth/variational_stereo_parameters.hpp>
#include <imp/cucore/cu_image_gpu.cuh>
#include <imp/cuimgproc/cu_image_filter.cuh>
#include <imp/cuimgproc/cu_image_transform.cuh>
#include <imp/cucore/cu_utils.hpp>
#include <imp/cucore/cu_texture.cuh>
#include <imp/cucore/cu_math.cuh>

#include <imp/io/opencv_bridge.hpp>

#include "cu_k_warped_gradients.cuh"
#include "cu_k_stereo_ctf_warping_level_huber_l1.cuh"

namespace imp {
namespace cu {


//------------------------------------------------------------------------------
StereoCtFWarpingLevelHuberL1::~StereoCtFWarpingLevelHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
StereoCtFWarpingLevelHuberL1::StereoCtFWarpingLevelHuberL1(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level)
  : StereoCtFWarpingLevel(params, size, level)
{
  u_.reset(new Image(size));
  u_prev_.reset(new Image(size));
  u0_.reset(new Image(size));
  pu_.reset(new Dual(size));
  ix_.reset(new Image(size));
  it_.reset(new Image(size));

  // and its textures
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelHuberL1::init(const StereoCtFWarpingLevel& rhs)
{
  const StereoCtFWarpingLevelHuberL1* from =
      dynamic_cast<const StereoCtFWarpingLevelHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(from->u0_.get(), from->u_.get());
    imp::cu::resample(u_.get(), from->u0_.get(), imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(u_.get(), from->u_.get(), imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(pu_.get(), from->pu_.get(), imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void StereoCtFWarpingLevelHuberL1::solve(std::vector<ImagePtr> images)
{
  std::cout << "StereoCtFWarpingLevelPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO

  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);
  u_->copyTo(*u_prev_);
  Fragmentation<16,16> frag(size_);

  // constants
  const float L = std::sqrt(8.f);
  const float tau = 1.f/L;
  const float sigma = 1.f/L;
  float lin_step = 0.5f;

  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    if (false && params_->ctf.apply_median_filter)
    {
      imp::cu::filterMedian3x3(u0_.get(), u_.get());
    }
    else
    {
      u_->copyTo(*u0_);
    }

    // compute warped spatial and temporal gradients
    k_warpedGradients
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (ix_->data(), it_->data(), ix_->stride(), ix_->width(), ix_->height(),
             *i1_tex_, *i2_tex_, *u0_tex_);

    if (params_->verbose > 10)
    {
      imp::cu::ocvBridgeShow("ix", *ix_, true);
      imp::cu::ocvBridgeShow("it", *it_, true);
    }

    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // dual kernel
      k_dualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(),
               size_.width(), size_.height(),
               params_->eps_u, sigma,
               *u_prev_tex_, *pu_tex_);

      // and primal kernel
      k_primalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               params_->lambda, tau, lin_step,
               *u_tex_, *u0_tex_, *pu_tex_, *ix_tex_, *it_tex_);

      if (params_->verbose > 5 && iter % 50)
      {
        imp::cu::ocvBridgeShow("current disp", *u_, true);
        imp::cu::ocvBridgeShow("current i0", *images.at(0), true);
        cv::waitKey(1);
      }

    } // iters
//    lin_step /= 1.2f;

  } // warps
  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

